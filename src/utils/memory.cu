
#include <hip/hip_runtime.h>
#include <stdint.h>

#ifdef __USE_CUDA__
void *calloc2(uint64_t count, uint64_t size) {
    char *a;
    cudaMallocManaged(&a, count*size);
#pragma omp parallel for
    for (uint64_t i = 0; i < count; i++) {
        memset(a+ i*size, 0, size);
    }
    return a;
}

void *malloc2(uint64_t size) {
    char *a;
    cudaMallocManaged(&a, size);
    return a;
}

void free2(void *ptr) { cudaFree(ptr); }
#endif
