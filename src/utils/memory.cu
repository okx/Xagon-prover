
#include <hip/hip_runtime.h>
#include <stdint.h>

#ifndef __USE_CUDA__
void *calloc_zkevm(uint64_t count, uint64_t size) {
    char *a;
    hipMallocManaged(&a, count*size);
#pragma omp parallel for
    for (uint64_t i = 0; i < count; i++) {
        memset(a+ i*size, 0, size);
    }
    return a;
}

void *malloc_zkevm(uint64_t size) {
    char *a;
    hipMallocManaged(&a, size);
    return a;
}

void free_zkevm(void *ptr) { hipFree(ptr); }
#endif
