#include "hip/hip_runtime.h"
#include "zklog.hpp"
#include <inttypes.h>

//#if defined(__USE_CUDA__) && defined(ENABLE_EXPERIMENTAL_CODE) && defined(__YXQ__)

#include "chelpers_steps_pack.cuh"
#include "goldilocks_cubic_extension.cuh"
#include "cuda_utils.cuh"
#include "cuda_utils.hpp"

void CHelpersStepsPackGPU::prepareGPU(StarkInfo &starkInfo, StepsParams &params, ParserArgs &parserArgs, ParserParams &parserParams) {

    Goldilocks::Element challenges[params.challenges.degree()*FIELD_EXTENSION*nrowsPack];
    Goldilocks::Element challenges_ops[params.challenges.degree()*FIELD_EXTENSION*nrowsPack];
    for(uint64_t i = 0; i < params.challenges.degree(); ++i) {
        for(uint64_t j = 0; j < nrowsPack; ++j) {
            challenges[(i*FIELD_EXTENSION)*nrowsPack + j] = params.challenges[i][0];
            challenges[(i*FIELD_EXTENSION + 1)*nrowsPack + j] = params.challenges[i][1];
            challenges[(i*FIELD_EXTENSION + 2)*nrowsPack + j] = params.challenges[i][2];
            challenges_ops[(i*FIELD_EXTENSION)*nrowsPack + j] = params.challenges[i][0] + params.challenges[i][1];
            challenges_ops[(i*FIELD_EXTENSION + 1)*nrowsPack + j] = params.challenges[i][0] + params.challenges[i][2];
            challenges_ops[(i*FIELD_EXTENSION + 2)*nrowsPack + j] = params.challenges[i][1] + params.challenges[i][2];
        }
    }

    Goldilocks::Element numbers_[parserParams.nNumbers*nrowsPack];
    for(uint64_t i = 0; i < parserParams.nNumbers; ++i) {
        for(uint64_t j = 0; j < nrowsPack; ++j) {
            numbers_[i*nrowsPack + j] = Goldilocks::fromU64(numbers[i]);
        }
    }

    Goldilocks::Element publics[starkInfo.nPublics*nrowsPack];
    for(uint64_t i = 0; i < starkInfo.nPublics; ++i) {
        for(uint64_t j = 0; j < nrowsPack; ++j) {
            publics[i*nrowsPack + j] = params.publicInputs[i];
        }
    }

    Goldilocks::Element evals[params.evals.degree()*FIELD_EXTENSION*nrowsPack];
    for(uint64_t i = 0; i < params.evals.degree(); ++i) {
        for(uint64_t j = 0; j < nrowsPack; ++j) {
            evals[(i*FIELD_EXTENSION)*nrowsPack + j] = params.evals[i][0];
            evals[(i*FIELD_EXTENSION + 1)*nrowsPack + j] = params.evals[i][1];
            evals[(i*FIELD_EXTENSION + 2)*nrowsPack + j] = params.evals[i][2];
        }
    }

    CHECKCUDAERR(hipMalloc(&ops_d, parserArgs.nOps * sizeof(uint8)));
    CHECKCUDAERR(hipMemcpy(ops_d, parserArgs.ops, parserArgs.nOps * sizeof(uint8), hipMemcpyHostToDevice));

    CHECKCUDAERR(hipMalloc(&arg_d, parserArgs.nArgs * sizeof(uint16_t)));
    CHECKCUDAERR(hipMemcpy(arg_d, parserArgs.args, parserArgs.nArgs * sizeof(uint16_t), hipMemcpyHostToDevice));

    CHECKCUDAERR(hipMalloc(&challenges_d, params.challenges.degree()*FIELD_EXTENSION*nrowsPack * sizeof(uint64_t)));
    CHECKCUDAERR(hipMemcpy(challenges_d, challenges, params.challenges.degree()*FIELD_EXTENSION*nrowsPack * sizeof(uint64_t), hipMemcpyHostToDevice));

    CHECKCUDAERR(hipMalloc(&challenges_ops_d, params.challenges.degree()*FIELD_EXTENSION*nrowsPack * sizeof(uint64_t)));
    CHECKCUDAERR(hipMemcpy(challenges_ops_d, challenges_ops, params.challenges.degree()*FIELD_EXTENSION*nrowsPack * sizeof(uint64_t), hipMemcpyHostToDevice));

    CHECKCUDAERR(hipMalloc(&numbers_d, parserParams.nNumbers*nrowsPack * sizeof(uint64_t)));
    CHECKCUDAERR(hipMemcpy(numbers_d, numbers, parserParams.nNumbers*nrowsPack * sizeof(uint64_t), hipMemcpyHostToDevice));

    CHECKCUDAERR(hipMalloc(&publics_d, starkInfo.nPublics*nrowsPack * sizeof(uint64_t)));
    CHECKCUDAERR(hipMemcpy(publics_d, publics, starkInfo.nPublics*nrowsPack * sizeof(uint64_t), hipMemcpyHostToDevice));

    CHECKCUDAERR(hipMalloc(&evals_d, params.evals.degree()*FIELD_EXTENSION*nrowsPack * sizeof(uint64_t)));
    CHECKCUDAERR(hipMemcpy(evals_d, evals, params.challenges.degree()*FIELD_EXTENSION*nrowsPack * sizeof(uint64_t), hipMemcpyHostToDevice));
}

void CHelpersStepsPackGPU::calculateExpressions(StarkInfo &starkInfo, StepsParams &params, ParserArgs &parserArgs, ParserParams &parserParams) {




    bool domainExtended = parserParams.stage > 3 ? true : false;
    uint64_t domainSize = domainExtended ? 1 << starkInfo.starkStruct.nBitsExt : 1 << starkInfo.starkStruct.nBits;
    calculateExpressionsRowsGPU(starkInfo, params, parserArgs, parserParams, 0, domainSize);
}

void CHelpersStepsPackGPU::calculateExpressionsRowsGPU(StarkInfo &starkInfo, StepsParams &params, ParserArgs &parserArgs, ParserParams &parserParams,
    uint64_t rowIni, uint64_t rowEnd){

    bool domainExtended = parserParams.stage > 3 ? true : false;
    uint64_t domainSize = domainExtended ? 1 << starkInfo.starkStruct.nBitsExt : 1 << starkInfo.starkStruct.nBits;
    uint8_t *ops = &parserArgs.ops[parserParams.opsOffset];
    uint16_t *args = &parserArgs.args[parserParams.argsOffset];
    uint64_t *numbers = &parserArgs.numbers[parserParams.numbersOffset];
    uint8_t *storePol = &parserArgs.storePols[parserParams.storePolsOffset];

    if(rowEnd < rowIni || rowEnd > domainSize) {
        zklog.info("Invalid range for rowIni and rowEnd");
        exitProcess();
    }
    if((rowEnd -rowIni) % nrowsPack != 0) {
       nrowsPack = 1;
    }

    setBufferTInfo(starkInfo, parserParams.stage);

    Goldilocks::Element bufferT_[2*nCols*nrowsPack];



}

__global__ void pack_kernel() {

}

//#endif