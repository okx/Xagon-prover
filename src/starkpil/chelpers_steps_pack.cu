#include "hip/hip_runtime.h"
#include "zklog.hpp"
#include <inttypes.h>

#if defined(__USE_CUDA__) && defined(ENABLE_EXPERIMENTAL_CODE)

#include "chelpers_steps_pack.cuh"
#include "goldilocks_cubic_extension.cuh"
#include "cuda_utils.cuh"
#include "cuda_utils.hpp"

void CHelpersStepsPackGPU::prepareGPU(StarkInfo &starkInfo, StepsParams &params, ParserArgs &parserArgs, ParserParams &parserParams) {

    Goldilocks::Element challenges[params.challenges.degree()*FIELD_EXTENSION*nrowsPack];
    Goldilocks::Element challenges_ops[params.challenges.degree()*FIELD_EXTENSION*nrowsPack];
    for(uint64_t i = 0; i < params.challenges.degree(); ++i) {
        for(uint64_t j = 0; j < nrowsPack; ++j) {
            challenges[(i*FIELD_EXTENSION)*nrowsPack + j] = params.challenges[i][0];
            challenges[(i*FIELD_EXTENSION + 1)*nrowsPack + j] = params.challenges[i][1];
            challenges[(i*FIELD_EXTENSION + 2)*nrowsPack + j] = params.challenges[i][2];
            challenges_ops[(i*FIELD_EXTENSION)*nrowsPack + j] = params.challenges[i][0] + params.challenges[i][1];
            challenges_ops[(i*FIELD_EXTENSION + 1)*nrowsPack + j] = params.challenges[i][0] + params.challenges[i][2];
            challenges_ops[(i*FIELD_EXTENSION + 2)*nrowsPack + j] = params.challenges[i][1] + params.challenges[i][2];
        }
    }

    Goldilocks::Element numbers_[parserParams.nNumbers*nrowsPack];
    for(uint64_t i = 0; i < parserParams.nNumbers; ++i) {
        for(uint64_t j = 0; j < nrowsPack; ++j) {
            numbers_[i*nrowsPack + j] = Goldilocks::fromU64(parserArgs.numbers[parserParams.numbersOffset+i]);
        }
    }

    Goldilocks::Element publics[starkInfo.nPublics*nrowsPack];
    for(uint64_t i = 0; i < starkInfo.nPublics; ++i) {
        for(uint64_t j = 0; j < nrowsPack; ++j) {
            publics[i*nrowsPack + j] = params.publicInputs[i];
        }
    }

    Goldilocks::Element evals[params.evals.degree()*FIELD_EXTENSION*nrowsPack];
    for(uint64_t i = 0; i < params.evals.degree(); ++i) {
        for(uint64_t j = 0; j < nrowsPack; ++j) {
            evals[(i*FIELD_EXTENSION)*nrowsPack + j] = params.evals[i][0];
            evals[(i*FIELD_EXTENSION + 1)*nrowsPack + j] = params.evals[i][1];
            evals[(i*FIELD_EXTENSION + 2)*nrowsPack + j] = params.evals[i][2];
        }
    }

    CHECKCUDAERR(hipMalloc(&nColsStagesAcc_d, nColsStagesAcc.size() * sizeof(uint8_t)));
    CHECKCUDAERR(hipMemcpy(nColsStagesAcc_d, nColsStagesAcc.data(), nColsStagesAcc.size() * sizeof(uint8_t), hipMemcpyHostToDevice));

    CHECKCUDAERR(hipMalloc(&ops_d, parserArgs.nOps * sizeof(uint8_t)));
    CHECKCUDAERR(hipMemcpy(ops_d, parserArgs.ops, parserArgs.nOps * sizeof(uint8_t), hipMemcpyHostToDevice));

    CHECKCUDAERR(hipMalloc(&args_d, parserArgs.nArgs * sizeof(uint16_t)));
    CHECKCUDAERR(hipMemcpy(args_d, parserArgs.args, parserArgs.nArgs * sizeof(uint16_t), hipMemcpyHostToDevice));

    CHECKCUDAERR(hipMalloc(&challenges_d, params.challenges.degree()*FIELD_EXTENSION*nrowsPack * sizeof(uint64_t)));
    CHECKCUDAERR(hipMemcpy(challenges_d, challenges, params.challenges.degree()*FIELD_EXTENSION*nrowsPack * sizeof(uint64_t), hipMemcpyHostToDevice));

    CHECKCUDAERR(hipMalloc(&challenges_ops_d, params.challenges.degree()*FIELD_EXTENSION*nrowsPack * sizeof(uint64_t)));
    CHECKCUDAERR(hipMemcpy(challenges_ops_d, challenges_ops, params.challenges.degree()*FIELD_EXTENSION*nrowsPack * sizeof(uint64_t), hipMemcpyHostToDevice));

    CHECKCUDAERR(hipMalloc(&numbers_d, parserParams.nNumbers*nrowsPack * sizeof(uint64_t)));
    CHECKCUDAERR(hipMemcpy(numbers_d, numbers_, parserParams.nNumbers*nrowsPack * sizeof(uint64_t), hipMemcpyHostToDevice));

    CHECKCUDAERR(hipMalloc(&publics_d, starkInfo.nPublics*nrowsPack * sizeof(uint64_t)));
    CHECKCUDAERR(hipMemcpy(publics_d, publics, starkInfo.nPublics*nrowsPack * sizeof(uint64_t), hipMemcpyHostToDevice));

    CHECKCUDAERR(hipMalloc(&evals_d, params.evals.degree()*FIELD_EXTENSION*nrowsPack * sizeof(uint64_t)));
    CHECKCUDAERR(hipMemcpy(evals_d, evals, params.challenges.degree()*FIELD_EXTENSION*nrowsPack * sizeof(uint64_t), hipMemcpyHostToDevice));
}

void CHelpersStepsPackGPU::cleanupGPU() {
    hipFree(nColsStagesAcc_d);
    hipFree(ops_d);
    hipFree(args_d);
    hipFree(challenges_d);
    hipFree(challenges_ops_d);
    hipFree(numbers_d);
    hipFree(publics_d);
    hipFree(evals_d);
}

void CHelpersStepsPackGPU::calculateExpressions(StarkInfo &starkInfo, StepsParams &params, ParserArgs &parserArgs, ParserParams &parserParams) {

    setBufferTInfo(starkInfo, parserParams.stage);
    prepareGPU(starkInfo, params, parserArgs, parserParams);

    bool domainExtended = parserParams.stage > 3 ? true : false;
    uint64_t domainSize = domainExtended ? 1 << starkInfo.starkStruct.nBitsExt : 1 << starkInfo.starkStruct.nBits;
    calculateExpressionsRowsGPU(starkInfo, params, parserArgs, parserParams, 0, domainSize);
    cleanupGPU();
}

void CHelpersStepsPackGPU::calculateExpressionsRowsGPU(StarkInfo &starkInfo, StepsParams &params, ParserArgs &parserArgs, ParserParams &parserParams,
    uint64_t rowIni, uint64_t rowEnd){

    bool domainExtended = parserParams.stage > 3 ? true : false;
    uint64_t domainSize = domainExtended ? 1 << starkInfo.starkStruct.nBitsExt : 1 << starkInfo.starkStruct.nBits;
    uint8_t *storePol = &parserArgs.storePols[parserParams.storePolsOffset];

    if(rowEnd < rowIni || rowEnd > domainSize) {
        zklog.info("Invalid range for rowIni and rowEnd");
        exitProcess();
    }
    if((rowEnd -rowIni) % nrowsPack != 0) {
       nrowsPack = 1;
    }

    Goldilocks::Element bufferT_[2*nCols*nrowsPack];
    gl64_t *bufferT_d;
    CHECKCUDAERR(hipMalloc(&bufferT_d, 2*nCols*nrowsPack * sizeof(uint64_t)));

    gl64_t *tmp1_d;
    gl64_t *tmp3_d;
    CHECKCUDAERR(hipMalloc(&tmp1_d, parserParams.nTemp1*nrowsPack * sizeof(uint64_t)));
    CHECKCUDAERR(hipMalloc(&tmp3_d, parserParams.nTemp3*FIELD_EXTENSION*nrowsPack * sizeof(uint64_t)));

    for (uint64_t i = rowIni; i < rowEnd; i+= nrowsPack) {
        printf("rows:%lu\n", i);
        loadPolinomials(starkInfo, params, bufferT_, i, parserParams.stage, nrowsPack, domainExtended);
        CHECKCUDAERR(hipMemcpy(bufferT_d, bufferT_, 2*nCols*nrowsPack * sizeof(uint16_t), hipMemcpyHostToDevice));
        pack_kernel<<<1,1>>>(nrowsPack, parserParams.nOps, parserParams.nArgs, tmp1_d, tmp3_d, nColsStagesAcc_d, ops_d, args_d, bufferT_d, challenges_d, challenges_ops_d, numbers_d, publics_d, evals_d);
        CHECKCUDAERR(hipMemcpy(bufferT_, bufferT_d, 2*nCols*nrowsPack * sizeof(uint16_t), hipMemcpyDeviceToHost));
        storePolinomials(starkInfo, params, bufferT_, storePol, i, nrowsPack, domainExtended);
    }

    hipFree(bufferT_d);
    hipFree(tmp1_d);
    hipFree(tmp3_d);
}

__global__ void pack_kernel(uint64_t nrowsPack,
                            uint32_t nOps,
                            uint32_t nArgs,
                            gl64_t *tmp1,
                            gl64_t *tmp3,
                            uint64_t *nColsStagesAcc,
                            uint8_t *ops,
                            uint16_t *args,
                            gl64_t *bufferT_,
                            gl64_t *challenges,
                            gl64_t *challenges_ops,
                            gl64_t *numbers_,
                            gl64_t *publics,
                            gl64_t *evals)
{
    uint64_t i_args = 0;

    for (uint64_t kk = 0; kk < nOps; ++kk) {
        switch (ops[kk]) {
            case 0: {
                // COPY commit1 to commit1
                gl64_t::copy_pack(nrowsPack, &bufferT_[(nColsStagesAcc[args[i_args]] + args[i_args + 1]) * nrowsPack], &bufferT_[(nColsStagesAcc[args[i_args + 2]] + args[i_args + 3]) * nrowsPack]);
                i_args += 4;
                break;
            }
            case 1: {
                // OPERATION WITH DEST: commit1 - SRC0: commit1 - SRC1: commit1
                gl64_t::op_pack(nrowsPack, args[i_args], &bufferT_[(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * nrowsPack], &bufferT_[(nColsStagesAcc[args[i_args + 3]] + args[i_args + 4]) * nrowsPack], &bufferT_[(nColsStagesAcc[args[i_args + 5]] + args[i_args + 6]) * nrowsPack]);
                i_args += 7;
                break;
            }
            case 2: {
                // OPERATION WITH DEST: commit1 - SRC0: commit1 - SRC1: tmp1
                gl64_t::op_pack(nrowsPack, args[i_args], &bufferT_[(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * nrowsPack], &bufferT_[(nColsStagesAcc[args[i_args + 3]] + args[i_args + 4]) * nrowsPack], &tmp1[args[i_args + 5] * nrowsPack]);
                i_args += 6;
                break;
            }
            case 3: {
                // OPERATION WITH DEST: commit1 - SRC0: commit1 - SRC1: public
                gl64_t::op_pack(nrowsPack, args[i_args], &bufferT_[(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * nrowsPack], &bufferT_[(nColsStagesAcc[args[i_args + 3]] + args[i_args + 4]) * nrowsPack], &publics[args[i_args + 5] * nrowsPack]);
                i_args += 6;
                break;
            }
            case 4: {
                // OPERATION WITH DEST: commit1 - SRC0: commit1 - SRC1: number
                gl64_t::op_pack(nrowsPack, args[i_args], &bufferT_[(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * nrowsPack], &bufferT_[(nColsStagesAcc[args[i_args + 3]] + args[i_args + 4]) * nrowsPack], &numbers_[args[i_args + 5]*nrowsPack]);
                i_args += 6;
                break;
            }
            case 5: {
                // COPY tmp1 to commit1
                gl64_t::copy_pack(nrowsPack, &bufferT_[(nColsStagesAcc[args[i_args]] + args[i_args + 1]) * nrowsPack], &tmp1[args[i_args + 2] * nrowsPack]);
                i_args += 3;
                break;
            }
            case 6: {
                // OPERATION WITH DEST: commit1 - SRC0: tmp1 - SRC1: tmp1
                gl64_t::op_pack(nrowsPack, args[i_args], &bufferT_[(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * nrowsPack], &tmp1[args[i_args + 3] * nrowsPack], &tmp1[args[i_args + 4] * nrowsPack]);
                i_args += 5;
                break;
            }
            case 7: {
                // OPERATION WITH DEST: commit1 - SRC0: tmp1 - SRC1: public
                gl64_t::op_pack(nrowsPack, args[i_args], &bufferT_[(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * nrowsPack], &tmp1[args[i_args + 3] * nrowsPack], &publics[args[i_args + 4] * nrowsPack]);
                i_args += 5;
                break;
            }
            case 8: {
                // OPERATION WITH DEST: commit1 - SRC0: tmp1 - SRC1: number
                gl64_t::op_pack(nrowsPack, args[i_args], &bufferT_[(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * nrowsPack], &tmp1[args[i_args + 3] * nrowsPack], &numbers_[args[i_args + 4]*nrowsPack]);
                i_args += 5;
                break;
            }
            case 9: {
                // COPY public to commit1
                gl64_t::copy_pack(nrowsPack, &bufferT_[(nColsStagesAcc[args[i_args]] + args[i_args + 1]) * nrowsPack], &publics[args[i_args + 2] * nrowsPack]);
                i_args += 3;
                break;
            }
            case 10: {
                // OPERATION WITH DEST: commit1 - SRC0: public - SRC1: public
                gl64_t::op_pack(nrowsPack, args[i_args], &bufferT_[(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * nrowsPack], &publics[args[i_args + 3] * nrowsPack], &publics[args[i_args + 4] * nrowsPack]);
                i_args += 5;
                break;
            }
            case 11: {
                // OPERATION WITH DEST: commit1 - SRC0: public - SRC1: number
                gl64_t::op_pack(nrowsPack, args[i_args], &bufferT_[(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * nrowsPack], &publics[args[i_args + 3] * nrowsPack], &numbers_[args[i_args + 4]*nrowsPack]);
                i_args += 5;
                break;
            }
            case 12: {
                // COPY number to commit1
                gl64_t::copy_pack(nrowsPack, &bufferT_[(nColsStagesAcc[args[i_args]] + args[i_args + 1]) * nrowsPack], &numbers_[args[i_args + 2]*nrowsPack]);
                i_args += 3;
                break;
            }
            case 13: {
                // OPERATION WITH DEST: commit1 - SRC0: number - SRC1: number
                gl64_t::op_pack(nrowsPack, args[i_args], &bufferT_[(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * nrowsPack], &numbers_[args[i_args + 3]*nrowsPack], &numbers_[args[i_args + 4]*nrowsPack]);
                i_args += 5;
                break;
            }
            case 14: {
                // COPY commit1 to tmp1
                gl64_t::copy_pack(nrowsPack, &tmp1[args[i_args] * nrowsPack], &bufferT_[(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * nrowsPack]);
                i_args += 3;
                break;
            }
            case 15: {
                // OPERATION WITH DEST: tmp1 - SRC0: commit1 - SRC1: commit1
                gl64_t::op_pack(nrowsPack, args[i_args], &tmp1[args[i_args + 1] * nrowsPack], &bufferT_[(nColsStagesAcc[args[i_args + 2]] + args[i_args + 3]) * nrowsPack], &bufferT_[(nColsStagesAcc[args[i_args + 4]] + args[i_args + 5]) * nrowsPack]);
                i_args += 6;
                break;
            }
            case 16: {
                // OPERATION WITH DEST: tmp1 - SRC0: commit1 - SRC1: tmp1
                gl64_t::op_pack(nrowsPack, args[i_args], &tmp1[args[i_args + 1] * nrowsPack], &bufferT_[(nColsStagesAcc[args[i_args + 2]] + args[i_args + 3]) * nrowsPack], &tmp1[args[i_args + 4] * nrowsPack]);
                i_args += 5;
                break;
            }
            case 17: {
                // OPERATION WITH DEST: tmp1 - SRC0: commit1 - SRC1: public
                gl64_t::op_pack(nrowsPack, args[i_args], &tmp1[args[i_args + 1] * nrowsPack], &bufferT_[(nColsStagesAcc[args[i_args + 2]] + args[i_args + 3]) * nrowsPack], &publics[args[i_args + 4] * nrowsPack]);
                i_args += 5;
                break;
            }
            case 18: {
                // OPERATION WITH DEST: tmp1 - SRC0: commit1 - SRC1: number
                gl64_t::op_pack(nrowsPack, args[i_args], &tmp1[args[i_args + 1] * nrowsPack], &bufferT_[(nColsStagesAcc[args[i_args + 2]] + args[i_args + 3]) * nrowsPack], &numbers_[args[i_args + 4]*nrowsPack]);
                i_args += 5;
                break;
            }
            case 19: {
                // COPY tmp1 to tmp1
                gl64_t::copy_pack(nrowsPack, &tmp1[args[i_args] * nrowsPack], &tmp1[args[i_args + 1] * nrowsPack]);
                i_args += 2;
                break;
            }
            case 20: {
                // OPERATION WITH DEST: tmp1 - SRC0: tmp1 - SRC1: tmp1
                gl64_t::op_pack(nrowsPack, args[i_args], &tmp1[args[i_args + 1] * nrowsPack], &tmp1[args[i_args + 2] * nrowsPack], &tmp1[args[i_args + 3] * nrowsPack]);
                i_args += 4;
                break;
            }
            case 21: {
                // OPERATION WITH DEST: tmp1 - SRC0: tmp1 - SRC1: public
                gl64_t::op_pack(nrowsPack, args[i_args], &tmp1[args[i_args + 1] * nrowsPack], &tmp1[args[i_args + 2] * nrowsPack], &publics[args[i_args + 3] * nrowsPack]);
                i_args += 4;
                break;
            }
            case 22: {
                // OPERATION WITH DEST: tmp1 - SRC0: tmp1 - SRC1: number
                gl64_t::op_pack(nrowsPack, args[i_args], &tmp1[args[i_args + 1] * nrowsPack], &tmp1[args[i_args + 2] * nrowsPack], &numbers_[args[i_args + 3]*nrowsPack]);
                i_args += 4;
                break;
            }
            case 23: {
                // COPY public to tmp1
                gl64_t::copy_pack(nrowsPack, &tmp1[args[i_args] * nrowsPack], &publics[args[i_args + 1] * nrowsPack]);
                i_args += 2;
                break;
            }
            case 24: {
                // OPERATION WITH DEST: tmp1 - SRC0: public - SRC1: public
                gl64_t::op_pack(nrowsPack, args[i_args], &tmp1[args[i_args + 1] * nrowsPack], &publics[args[i_args + 2] * nrowsPack], &publics[args[i_args + 3] * nrowsPack]);
                i_args += 4;
                break;
            }
            case 25: {
                // OPERATION WITH DEST: tmp1 - SRC0: public - SRC1: number
                gl64_t::op_pack(nrowsPack, args[i_args], &tmp1[args[i_args + 1] * nrowsPack], &publics[args[i_args + 2] * nrowsPack], &numbers_[args[i_args + 3]*nrowsPack]);
                i_args += 4;
                break;
            }
            case 26: {
                // COPY number to tmp1
                gl64_t::copy_pack(nrowsPack, &tmp1[args[i_args] * nrowsPack], &numbers_[args[i_args + 1]*nrowsPack]);
                i_args += 2;
                break;
            }
            case 27: {
                // OPERATION WITH DEST: tmp1 - SRC0: number - SRC1: number
                gl64_t::op_pack(nrowsPack, args[i_args], &tmp1[args[i_args + 1] * nrowsPack], &numbers_[args[i_args + 2]*nrowsPack], &numbers_[args[i_args + 3]*nrowsPack]);
                i_args += 4;
                break;
            }
            case 28: {
                // OPERATION WITH DEST: commit3 - SRC0: commit3 - SRC1: commit1
                Goldilocks3GPU::op_31_pack(nrowsPack, args[i_args], &bufferT_[(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * nrowsPack], &bufferT_[(nColsStagesAcc[args[i_args + 3]] + args[i_args + 4]) * nrowsPack], &bufferT_[(nColsStagesAcc[args[i_args + 5]] + args[i_args + 6]) * nrowsPack]);
                i_args += 7;
                break;
            }
            case 29: {
                // OPERATION WITH DEST: commit3 - SRC0: commit3 - SRC1: tmp1
                Goldilocks3GPU::op_31_pack(nrowsPack, args[i_args], &bufferT_[(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * nrowsPack], &bufferT_[(nColsStagesAcc[args[i_args + 3]] + args[i_args + 4]) * nrowsPack], &tmp1[args[i_args + 5] * nrowsPack]);
                i_args += 6;
                break;
            }
            case 30: {
                // OPERATION WITH DEST: commit3 - SRC0: commit3 - SRC1: public
                Goldilocks3GPU::op_31_pack(nrowsPack, args[i_args], &bufferT_[(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * nrowsPack], &bufferT_[(nColsStagesAcc[args[i_args + 3]] + args[i_args + 4]) * nrowsPack], &publics[args[i_args + 5] * nrowsPack]);
                i_args += 6;
                break;
            }
            case 31: {
                // OPERATION WITH DEST: commit3 - SRC0: commit3 - SRC1: number
                Goldilocks3GPU::op_31_pack(nrowsPack, args[i_args], &bufferT_[(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * nrowsPack], &bufferT_[(nColsStagesAcc[args[i_args + 3]] + args[i_args + 4]) * nrowsPack], &numbers_[args[i_args + 5]*nrowsPack]);
                i_args += 6;
                break;
            }
            case 32: {
                // OPERATION WITH DEST: commit3 - SRC0: tmp3 - SRC1: commit1
                Goldilocks3GPU::op_31_pack(nrowsPack, args[i_args], &bufferT_[(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * nrowsPack], &tmp3[args[i_args + 3] * nrowsPack * FIELD_EXTENSION], &bufferT_[(nColsStagesAcc[args[i_args + 4]] + args[i_args + 5]) * nrowsPack]);
                i_args += 6;
                break;
            }
            case 33: {
                // OPERATION WITH DEST: commit3 - SRC0: tmp3 - SRC1: tmp1
                Goldilocks3GPU::op_31_pack(nrowsPack, args[i_args], &bufferT_[(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * nrowsPack], &tmp3[args[i_args + 3] * nrowsPack * FIELD_EXTENSION], &tmp1[args[i_args + 4] * nrowsPack]);
                i_args += 5;
                break;
            }
            case 34: {
                // OPERATION WITH DEST: commit3 - SRC0: tmp3 - SRC1: public
                Goldilocks3GPU::op_31_pack(nrowsPack, args[i_args], &bufferT_[(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * nrowsPack], &tmp3[args[i_args + 3] * nrowsPack * FIELD_EXTENSION], &publics[args[i_args + 4] * nrowsPack]);
                i_args += 5;
                break;
            }
            case 35: {
                // OPERATION WITH DEST: commit3 - SRC0: tmp3 - SRC1: number
                Goldilocks3GPU::op_31_pack(nrowsPack, args[i_args], &bufferT_[(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * nrowsPack], &tmp3[args[i_args + 3] * nrowsPack * FIELD_EXTENSION], &numbers_[args[i_args + 4]*nrowsPack]);
                i_args += 5;
                break;
            }
            case 36: {
                // OPERATION WITH DEST: commit3 - SRC0: challenge - SRC1: commit1
                Goldilocks3GPU::op_31_pack(nrowsPack, args[i_args], &bufferT_[(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * nrowsPack], &challenges[args[i_args + 3]*FIELD_EXTENSION*nrowsPack], &bufferT_[(nColsStagesAcc[args[i_args + 4]] + args[i_args + 5]) * nrowsPack]);
                i_args += 6;
                break;
            }
            case 37: {
                // OPERATION WITH DEST: commit3 - SRC0: challenge - SRC1: tmp1
                Goldilocks3GPU::op_31_pack(nrowsPack, args[i_args], &bufferT_[(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * nrowsPack], &challenges[args[i_args + 3]*FIELD_EXTENSION*nrowsPack], &tmp1[args[i_args + 4] * nrowsPack]);
                i_args += 5;
                break;
            }
            case 38: {
                // OPERATION WITH DEST: commit3 - SRC0: challenge - SRC1: public
                Goldilocks3GPU::op_31_pack(nrowsPack, args[i_args], &bufferT_[(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * nrowsPack], &challenges[args[i_args + 3]*FIELD_EXTENSION*nrowsPack], &publics[args[i_args + 4] * nrowsPack]);
                i_args += 5;
                break;
            }
            case 39: {
                // OPERATION WITH DEST: commit3 - SRC0: challenge - SRC1: number
                Goldilocks3GPU::op_31_pack(nrowsPack, args[i_args], &bufferT_[(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * nrowsPack], &challenges[args[i_args + 3]*FIELD_EXTENSION*nrowsPack], &numbers_[args[i_args + 4]*nrowsPack]);
                i_args += 5;
                break;
            }
            case 40: {
                // COPY commit3 to commit3
                Goldilocks3GPU::copy_pack(nrowsPack, &bufferT_[(nColsStagesAcc[args[i_args]] + args[i_args + 1]) * nrowsPack], &bufferT_[(nColsStagesAcc[args[i_args + 2]] + args[i_args + 3]) * nrowsPack]);
                i_args += 4;
                break;
            }
            case 41: {
                // OPERATION WITH DEST: commit3 - SRC0: commit3 - SRC1: commit3
                Goldilocks3GPU::op_pack(nrowsPack, args[i_args], &bufferT_[(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * nrowsPack], &bufferT_[(nColsStagesAcc[args[i_args + 3]] + args[i_args + 4]) * nrowsPack], &bufferT_[(nColsStagesAcc[args[i_args + 5]] + args[i_args + 6]) * nrowsPack]);
                i_args += 7;
                break;
            }
            case 42: {
                // OPERATION WITH DEST: commit3 - SRC0: commit3 - SRC1: tmp3
                Goldilocks3GPU::op_pack(nrowsPack, args[i_args], &bufferT_[(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * nrowsPack], &bufferT_[(nColsStagesAcc[args[i_args + 3]] + args[i_args + 4]) * nrowsPack], &tmp3[args[i_args + 5] * nrowsPack * FIELD_EXTENSION]);
                i_args += 6;
                break;
            }
            case 43: {
                // MULTIPLICATION WITH DEST: commit3 - SRC0: commit3 - SRC1: challenge
                Goldilocks3GPU::mul_pack(nrowsPack, &bufferT_[(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * nrowsPack], &bufferT_[(nColsStagesAcc[args[i_args + 3]] + args[i_args + 4]) * nrowsPack], &challenges[args[i_args + 5]*FIELD_EXTENSION*nrowsPack], &challenges_ops[args[i_args + 5]*FIELD_EXTENSION*nrowsPack]);
                i_args += 6;
                break;
            }
            case 44: {
                // OPERATION WITH DEST: commit3 - SRC0: commit3 - SRC1: challenge
                Goldilocks3GPU::op_pack(nrowsPack, args[i_args], &bufferT_[(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * nrowsPack], &bufferT_[(nColsStagesAcc[args[i_args + 3]] + args[i_args + 4]) * nrowsPack], &challenges[args[i_args + 5]*FIELD_EXTENSION*nrowsPack]);
                i_args += 6;
                break;
            }
            case 45: {
                // COPY tmp3 to commit3
                Goldilocks3GPU::copy_pack(nrowsPack, &bufferT_[(nColsStagesAcc[args[i_args]] + args[i_args + 1]) * nrowsPack], &tmp3[args[i_args + 2] * nrowsPack * FIELD_EXTENSION]);
                i_args += 3;
                break;
            }
            case 46: {
                // OPERATION WITH DEST: commit3 - SRC0: tmp3 - SRC1: tmp3
                Goldilocks3GPU::op_pack(nrowsPack, args[i_args], &bufferT_[(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * nrowsPack], &tmp3[args[i_args + 3] * nrowsPack * FIELD_EXTENSION], &tmp3[args[i_args + 4] * nrowsPack * FIELD_EXTENSION]);
                i_args += 5;
                break;
            }
            case 47: {
                // MULTIPLICATION WITH DEST: commit3 - SRC0: tmp3 - SRC1: challenge
                Goldilocks3GPU::mul_pack(nrowsPack, &bufferT_[(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * nrowsPack], &tmp3[args[i_args + 3] * nrowsPack * FIELD_EXTENSION], &challenges[args[i_args + 4]*FIELD_EXTENSION*nrowsPack], &challenges_ops[args[i_args + 4]*FIELD_EXTENSION*nrowsPack]);
                i_args += 5;
                break;
            }
            case 48: {
                // OPERATION WITH DEST: commit3 - SRC0: tmp3 - SRC1: challenge
                Goldilocks3GPU::op_pack(nrowsPack, args[i_args], &bufferT_[(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * nrowsPack], &tmp3[args[i_args + 3] * nrowsPack * FIELD_EXTENSION], &challenges[args[i_args + 4]*FIELD_EXTENSION*nrowsPack]);
                i_args += 5;
                break;
            }
            case 49: {
                // MULTIPLICATION WITH DEST: commit3 - SRC0: challenge - SRC1: challenge
                Goldilocks3GPU::mul_pack(nrowsPack, &bufferT_[(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * nrowsPack], &challenges[args[i_args + 3]*FIELD_EXTENSION*nrowsPack], &challenges[args[i_args + 4]*FIELD_EXTENSION*nrowsPack], &challenges_ops[args[i_args + 4]*FIELD_EXTENSION*nrowsPack]);
                i_args += 5;
                break;
            }
            case 50: {
                // OPERATION WITH DEST: commit3 - SRC0: challenge - SRC1: challenge
                Goldilocks3GPU::op_pack(nrowsPack, args[i_args], &bufferT_[(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * nrowsPack], &challenges[args[i_args + 3]*FIELD_EXTENSION*nrowsPack], &challenges[args[i_args + 4]*FIELD_EXTENSION*nrowsPack]);
                i_args += 5;
                break;
            }
            case 51: {
                // OPERATION WITH DEST: tmp3 - SRC0: commit3 - SRC1: commit1
                Goldilocks3GPU::op_31_pack(nrowsPack, args[i_args], &tmp3[args[i_args + 1] * nrowsPack * FIELD_EXTENSION], &bufferT_[(nColsStagesAcc[args[i_args + 2]] + args[i_args + 3]) * nrowsPack], &bufferT_[(nColsStagesAcc[args[i_args + 4]] + args[i_args + 5]) * nrowsPack]);
                i_args += 6;
                break;
            }
            case 52: {
                // OPERATION WITH DEST: tmp3 - SRC0: commit3 - SRC1: tmp1
                Goldilocks3GPU::op_31_pack(nrowsPack, args[i_args], &tmp3[args[i_args + 1] * nrowsPack * FIELD_EXTENSION], &bufferT_[(nColsStagesAcc[args[i_args + 2]] + args[i_args + 3]) * nrowsPack], &tmp1[args[i_args + 4] * nrowsPack]);
                i_args += 5;
                break;
            }
            case 53: {
                // OPERATION WITH DEST: tmp3 - SRC0: commit3 - SRC1: public
                Goldilocks3GPU::op_31_pack(nrowsPack, args[i_args], &tmp3[args[i_args + 1] * nrowsPack * FIELD_EXTENSION], &bufferT_[(nColsStagesAcc[args[i_args + 2]] + args[i_args + 3]) * nrowsPack], &publics[args[i_args + 4] * nrowsPack]);
                i_args += 5;
                break;
            }
            case 54: {
                // OPERATION WITH DEST: tmp3 - SRC0: commit3 - SRC1: number
                Goldilocks3GPU::op_31_pack(nrowsPack, args[i_args], &tmp3[args[i_args + 1] * nrowsPack * FIELD_EXTENSION], &bufferT_[(nColsStagesAcc[args[i_args + 2]] + args[i_args + 3]) * nrowsPack], &numbers_[args[i_args + 4]*nrowsPack]);
                i_args += 5;
                break;
            }
            case 55: {
                // OPERATION WITH DEST: tmp3 - SRC0: tmp3 - SRC1: commit1
                Goldilocks3GPU::op_31_pack(nrowsPack, args[i_args], &tmp3[args[i_args + 1] * nrowsPack * FIELD_EXTENSION], &tmp3[args[i_args + 2] * nrowsPack * FIELD_EXTENSION], &bufferT_[(nColsStagesAcc[args[i_args + 3]] + args[i_args + 4]) * nrowsPack]);
                i_args += 5;
                break;
            }
            case 56: {
                // OPERATION WITH DEST: tmp3 - SRC0: tmp3 - SRC1: tmp1
                Goldilocks3GPU::op_31_pack(nrowsPack, args[i_args], &tmp3[args[i_args + 1] * nrowsPack * FIELD_EXTENSION], &tmp3[args[i_args + 2] * nrowsPack * FIELD_EXTENSION], &tmp1[args[i_args + 3] * nrowsPack]);
                i_args += 4;
                break;
            }
            case 57: {
                // OPERATION WITH DEST: tmp3 - SRC0: tmp3 - SRC1: public
                Goldilocks3GPU::op_31_pack(nrowsPack, args[i_args], &tmp3[args[i_args + 1] * nrowsPack * FIELD_EXTENSION], &tmp3[args[i_args + 2] * nrowsPack * FIELD_EXTENSION], &publics[args[i_args + 3] * nrowsPack]);
                i_args += 4;
                break;
            }
            case 58: {
                // OPERATION WITH DEST: tmp3 - SRC0: tmp3 - SRC1: number
                Goldilocks3GPU::op_31_pack(nrowsPack, args[i_args], &tmp3[args[i_args + 1] * nrowsPack * FIELD_EXTENSION], &tmp3[args[i_args + 2] * nrowsPack * FIELD_EXTENSION], &numbers_[args[i_args + 3]*nrowsPack]);
                i_args += 4;
                break;
            }
            case 59: {
                // OPERATION WITH DEST: tmp3 - SRC0: challenge - SRC1: commit1
                Goldilocks3GPU::op_31_pack(nrowsPack, args[i_args], &tmp3[args[i_args + 1] * nrowsPack * FIELD_EXTENSION], &challenges[args[i_args + 2]*FIELD_EXTENSION*nrowsPack], &bufferT_[(nColsStagesAcc[args[i_args + 3]] + args[i_args + 4]) * nrowsPack]);
                i_args += 5;
                break;
            }
            case 60: {
                // OPERATION WITH DEST: tmp3 - SRC0: challenge - SRC1: tmp1
                Goldilocks3GPU::op_31_pack(nrowsPack, args[i_args], &tmp3[args[i_args + 1] * nrowsPack * FIELD_EXTENSION], &challenges[args[i_args + 2]*FIELD_EXTENSION*nrowsPack], &tmp1[args[i_args + 3] * nrowsPack]);
                i_args += 4;
                break;
            }
            case 61: {
                // OPERATION WITH DEST: tmp3 - SRC0: challenge - SRC1: public
                Goldilocks3GPU::op_31_pack(nrowsPack, args[i_args], &tmp3[args[i_args + 1] * nrowsPack * FIELD_EXTENSION], &challenges[args[i_args + 2]*FIELD_EXTENSION*nrowsPack], &publics[args[i_args + 3] * nrowsPack]);
                i_args += 4;
                break;
            }
            case 62: {
                // OPERATION WITH DEST: tmp3 - SRC0: challenge - SRC1: number
                Goldilocks3GPU::op_31_pack(nrowsPack, args[i_args], &tmp3[args[i_args + 1] * nrowsPack * FIELD_EXTENSION], &challenges[args[i_args + 2]*FIELD_EXTENSION*nrowsPack], &numbers_[args[i_args + 3]*nrowsPack]);
                i_args += 4;
                break;
            }
            case 63: {
                // COPY commit3 to tmp3
                Goldilocks3GPU::copy_pack(nrowsPack, &tmp3[args[i_args] * nrowsPack * FIELD_EXTENSION], &bufferT_[(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * nrowsPack]);
                i_args += 3;
                break;
            }
            case 64: {
                // OPERATION WITH DEST: tmp3 - SRC0: commit3 - SRC1: commit3
                Goldilocks3GPU::op_pack(nrowsPack, args[i_args], &tmp3[args[i_args + 1] * nrowsPack * FIELD_EXTENSION], &bufferT_[(nColsStagesAcc[args[i_args + 2]] + args[i_args + 3]) * nrowsPack], &bufferT_[(nColsStagesAcc[args[i_args + 4]] + args[i_args + 5]) * nrowsPack]);
                i_args += 6;
                break;
            }
            case 65: {
                // OPERATION WITH DEST: tmp3 - SRC0: commit3 - SRC1: tmp3
                Goldilocks3GPU::op_pack(nrowsPack, args[i_args], &tmp3[args[i_args + 1] * nrowsPack * FIELD_EXTENSION], &bufferT_[(nColsStagesAcc[args[i_args + 2]] + args[i_args + 3]) * nrowsPack], &tmp3[args[i_args + 4] * nrowsPack * FIELD_EXTENSION]);
                i_args += 5;
                break;
            }
            case 66: {
                // MULTIPLICATION WITH DEST: tmp3 - SRC0: commit3 - SRC1: challenge
                Goldilocks3GPU::mul_pack(nrowsPack, &tmp3[args[i_args + 1] * nrowsPack * FIELD_EXTENSION], &bufferT_[(nColsStagesAcc[args[i_args + 2]] + args[i_args + 3]) * nrowsPack], &challenges[args[i_args + 4]*FIELD_EXTENSION*nrowsPack], &challenges_ops[args[i_args + 4]*FIELD_EXTENSION*nrowsPack]);
                i_args += 5;
                break;
            }
            case 67: {
                // OPERATION WITH DEST: tmp3 - SRC0: commit3 - SRC1: challenge
                Goldilocks3GPU::op_pack(nrowsPack, args[i_args], &tmp3[args[i_args + 1] * nrowsPack * FIELD_EXTENSION], &bufferT_[(nColsStagesAcc[args[i_args + 2]] + args[i_args + 3]) * nrowsPack], &challenges[args[i_args + 4]*FIELD_EXTENSION*nrowsPack]);
                i_args += 5;
                break;
            }
            case 68: {
                // COPY tmp3 to tmp3
                Goldilocks3GPU::copy_pack(nrowsPack, &tmp3[args[i_args] * nrowsPack * FIELD_EXTENSION], &tmp3[args[i_args + 1] * nrowsPack * FIELD_EXTENSION]);
                i_args += 2;
                break;
            }
            case 69: {
                // OPERATION WITH DEST: tmp3 - SRC0: tmp3 - SRC1: tmp3
                Goldilocks3GPU::op_pack(nrowsPack, args[i_args], &tmp3[args[i_args + 1] * nrowsPack * FIELD_EXTENSION], &tmp3[args[i_args + 2] * nrowsPack * FIELD_EXTENSION], &tmp3[args[i_args + 3] * nrowsPack * FIELD_EXTENSION]);
                i_args += 4;
                break;
            }
            case 70: {
                // MULTIPLICATION WITH DEST: tmp3 - SRC0: tmp3 - SRC1: challenge
                Goldilocks3GPU::mul_pack(nrowsPack, &tmp3[args[i_args + 1] * nrowsPack * FIELD_EXTENSION], &tmp3[args[i_args + 2] * nrowsPack * FIELD_EXTENSION], &challenges[args[i_args + 3]*FIELD_EXTENSION*nrowsPack], &challenges_ops[args[i_args + 3]*FIELD_EXTENSION*nrowsPack]);
                i_args += 4;
                break;
            }
            case 71: {
                // OPERATION WITH DEST: tmp3 - SRC0: tmp3 - SRC1: challenge
                Goldilocks3GPU::op_pack(nrowsPack, args[i_args], &tmp3[args[i_args + 1] * nrowsPack * FIELD_EXTENSION], &tmp3[args[i_args + 2] * nrowsPack * FIELD_EXTENSION], &challenges[args[i_args + 3]*FIELD_EXTENSION*nrowsPack]);
                i_args += 4;
                break;
            }
            case 72: {
                // MULTIPLICATION WITH DEST: tmp3 - SRC0: challenge - SRC1: challenge
                Goldilocks3GPU::mul_pack(nrowsPack, &tmp3[args[i_args + 1] * nrowsPack * FIELD_EXTENSION], &challenges[args[i_args + 2]*FIELD_EXTENSION*nrowsPack], &challenges[args[i_args + 3]*FIELD_EXTENSION*nrowsPack], &challenges_ops[args[i_args + 3]*FIELD_EXTENSION*nrowsPack]);
                i_args += 4;
                break;
            }
            case 73: {
                // OPERATION WITH DEST: tmp3 - SRC0: challenge - SRC1: challenge
                Goldilocks3GPU::op_pack(nrowsPack, args[i_args], &tmp3[args[i_args + 1] * nrowsPack * FIELD_EXTENSION], &challenges[args[i_args + 2]*FIELD_EXTENSION*nrowsPack], &challenges[args[i_args + 3]*FIELD_EXTENSION*nrowsPack]);
                i_args += 4;
                break;
            }
            case 74: {
                // COPY eval to tmp3
                Goldilocks3GPU::copy_pack(nrowsPack, &tmp3[args[i_args] * nrowsPack * FIELD_EXTENSION], &evals[args[i_args + 1]*FIELD_EXTENSION*nrowsPack]);
                i_args += 2;
                break;
            }
            case 75: {
                // MULTIPLICATION WITH DEST: tmp3 - SRC0: eval - SRC1: challenge
                Goldilocks3GPU::mul_pack(nrowsPack, &tmp3[args[i_args + 1] * nrowsPack * FIELD_EXTENSION], &evals[args[i_args + 2]*FIELD_EXTENSION*nrowsPack], &challenges[args[i_args + 3]*FIELD_EXTENSION*nrowsPack], &challenges_ops[args[i_args + 3]*FIELD_EXTENSION*nrowsPack]);
                i_args += 4;
                break;
            }
            case 76: {
                // OPERATION WITH DEST: tmp3 - SRC0: challenge - SRC1: eval
                Goldilocks3GPU::op_pack(nrowsPack, args[i_args], &tmp3[args[i_args + 1] * nrowsPack * FIELD_EXTENSION], &challenges[args[i_args + 2]*FIELD_EXTENSION*nrowsPack], &evals[args[i_args + 3]*FIELD_EXTENSION*nrowsPack]);
                i_args += 4;
                break;
            }
            case 77: {
                // OPERATION WITH DEST: tmp3 - SRC0: tmp3 - SRC1: eval
                Goldilocks3GPU::op_pack(nrowsPack, args[i_args], &tmp3[args[i_args + 1] * nrowsPack * FIELD_EXTENSION], &tmp3[args[i_args + 2] * nrowsPack * FIELD_EXTENSION], &evals[args[i_args + 3]*FIELD_EXTENSION*nrowsPack]);
                i_args += 4;
                break;
            }
            case 78: {
                // OPERATION WITH DEST: tmp3 - SRC0: eval - SRC1: commit1
                Goldilocks3GPU::op_31_pack(nrowsPack, args[i_args], &tmp3[args[i_args + 1] * nrowsPack * FIELD_EXTENSION], &evals[args[i_args + 2]*FIELD_EXTENSION*nrowsPack], &bufferT_[(nColsStagesAcc[args[i_args + 3]] + args[i_args + 4]) * nrowsPack]);
                i_args += 5;
                break;
            }
            case 79: {
                // OPERATION WITH DEST: tmp3 - SRC0: commit3 - SRC1: eval
                Goldilocks3GPU::op_pack(nrowsPack, args[i_args], &tmp3[args[i_args + 1] * nrowsPack * FIELD_EXTENSION], &bufferT_[(nColsStagesAcc[args[i_args + 2]] + args[i_args + 3]) * nrowsPack], &evals[args[i_args + 4]*FIELD_EXTENSION*nrowsPack]);
                i_args += 5;
                break;
            }
            default: {
                assert(false);
            }
        }
    }

    assert(i_args == nArgs);

}

#endif