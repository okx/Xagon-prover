#include "hip/hip_runtime.h"
#include "zklog.hpp"
#include <inttypes.h>

#if defined(__USE_CUDA__) && defined(ENABLE_EXPERIMENTAL_CODE)

#include "chelpers_steps_pack.cuh"
#include "goldilocks_cubic_extension.cuh"
#include "cuda_utils.cuh"
#include "cuda_utils.hpp"
#include "timer.hpp"

bool writeDataToFile(const std::string& filename, const uint64_t* data, size_t size) {
    // 打开文件
    std::ofstream file(filename, std::ios::app);
    if (file.is_open()) {
        // 逐行写入数据
        for (size_t i = 0; i < size; i++) {
            file << (data[i] % 18446744069414584321) << std::endl;
        }
        // 关闭文件
        file.close();
        std::cout << "Data written to file successfully!" << std::endl;
        return true;
    } else {
        std::cerr << "Unable to open file." << std::endl;
        return false;
    }
}

const uint64_t MAX_U64 = 0xFFFFFFFFFFFFFFFF;

void CHelpersStepsPackGPU::prepareGPU(StarkInfo &starkInfo, StepsParams &params, ParserArgs &parserArgs, ParserParams &parserParams) {

    prepare(starkInfo, params, parserArgs, parserParams);

    nCudaThreads = 1<<15;
    domainExtended = parserParams.stage > 3 ? true : false;
    domainSize = domainExtended ? 1 << starkInfo.starkStruct.nBitsExt : 1 << starkInfo.starkStruct.nBits;
    subDomainSize = nrowsPack * nCudaThreads;
    nextStride = domainExtended ? 1 << (starkInfo.starkStruct.nBitsExt - starkInfo.starkStruct.nBits) : 1;

    nOps = parserParams.nOps;
    nArgs = parserParams.nArgs;
    nBufferT = 2*nCols*nrowsPack;
    nTemp1 = parserParams.nTemp1*nrowsPack;
    nTemp3 = parserParams.nTemp3*FIELD_EXTENSION*nrowsPack;

    printf("nCols:%lu\n", nCols);
    printf("nrowsPack:%lu\n", nrowsPack);
    printf("buffer:%lu\n", nBufferT);

    offsetsStagesGPU.resize(offsetsStages.size());
    uint64_t total_pols = 0;
    for (uint64_t s = 1; s < 11; s++) {
        if (s < 4 || (s == 4 && parserParams.stage != 4) || (s == 10 && domainExtended)) {
            printf("s=%lu, offsets=%lu\n", s, total_pols);
            offsetsStagesGPU[s] = total_pols;
            total_pols += nColsStages[s] * (nrowsPack * nCudaThreads + nextStride);
        } else {
            offsetsStagesGPU[s] = MAX_U64;
        }
    }

    printf("total_pols:%lu\n", total_pols);

    assert(sharedStorageSize == 0);
    ops_offset = sharedStorageSize;
    sharedStorageSize += nOps;

    args_offset = sharedStorageSize;
    sharedStorageSize += nArgs;

    offsetsStages_offset = sharedStorageSize;
    sharedStorageSize += offsetsStages.size();

    nColsStages_offset = sharedStorageSize;
    sharedStorageSize += nColsStages.size();

    nColsStagesAcc_offset = sharedStorageSize;
    sharedStorageSize += nColsStagesAcc.size();

    challenges_offset = sharedStorageSize;
    sharedStorageSize += challenges.size();

    challenges_ops_offset = sharedStorageSize;
    sharedStorageSize += challenges_ops.size();

    numbers_offset = sharedStorageSize;
    sharedStorageSize += numbers_.size();

    publics_offset = sharedStorageSize;
    sharedStorageSize += publics.size();

    evals_offset = sharedStorageSize;
    sharedStorageSize += evals.size();

    printf("sharedStorageSize:%lu\n", sharedStorageSize);

    CHECKCUDAERR(hipMalloc(&sharedStorage, sharedStorageSize * sizeof(uint64_t)));

    uint64_t *ops64 = (uint64_t *)malloc(nOps * sizeof(uint64_t));
    for (uint32_t i=0; i<nOps; i++) {
        ops64[i] = uint64_t(parserArgs.ops[parserParams.opsOffset+i]);
    }
    CHECKCUDAERR(hipMemcpy(sharedStorage+ops_offset, ops64, nOps * sizeof(uint64_t), hipMemcpyHostToDevice));
    uint64_t *args64 = (uint64_t *)malloc(nArgs * sizeof(uint64_t));
    for (uint32_t i=0; i<nArgs; i++) {
        args64[i] = uint64_t(parserArgs.args[parserParams.argsOffset+i]);
    }
    CHECKCUDAERR(hipMemcpy(sharedStorage+args_offset, args64, nArgs * sizeof(uint16_t), hipMemcpyHostToDevice));
    CHECKCUDAERR(hipMemcpy(sharedStorage+offsetsStages_offset, offsetsStagesGPU.data(), offsetsStagesGPU.size() * sizeof(uint64_t), hipMemcpyHostToDevice));
    CHECKCUDAERR(hipMemcpy(sharedStorage+nColsStages_offset, nColsStages.data(), nColsStages.size() * sizeof(uint64_t), hipMemcpyHostToDevice));
    CHECKCUDAERR(hipMemcpy(sharedStorage+nColsStagesAcc_offset, nColsStagesAcc.data(), nColsStagesAcc.size() * sizeof(uint64_t), hipMemcpyHostToDevice));
    CHECKCUDAERR(hipMemcpy(sharedStorage+challenges_offset, challenges.data(), challenges.size() * sizeof(uint64_t), hipMemcpyHostToDevice));
    CHECKCUDAERR(hipMemcpy(sharedStorage+challenges_ops_offset, challenges_ops.data(), challenges_ops.size() * sizeof(uint64_t), hipMemcpyHostToDevice));
    CHECKCUDAERR(hipMemcpy(sharedStorage+numbers_offset, numbers_.data(), numbers_.size() * sizeof(uint64_t), hipMemcpyHostToDevice));
    CHECKCUDAERR(hipMemcpy(sharedStorage+publics_offset, publics.data(), publics.size() * sizeof(uint64_t), hipMemcpyHostToDevice));
    CHECKCUDAERR(hipMemcpy(sharedStorage+evals_offset, evals.data(), evals.size() * sizeof(uint64_t), hipMemcpyHostToDevice));


    assert(exclusiveStorageSize==0);
    constPols_offset = exclusiveStorageSize;
    exclusiveStorageSize += starkInfo.nConstants * (subDomainSize + nextStride);

    x_offset = exclusiveStorageSize;
    exclusiveStorageSize += subDomainSize;

    zi_offset = exclusiveStorageSize;
    exclusiveStorageSize += subDomainSize;

    pols_offset = exclusiveStorageSize;
    exclusiveStorageSize += total_pols;

    xDivXSubXi_offset = exclusiveStorageSize;
    exclusiveStorageSize += 2 * subDomainSize * FIELD_EXTENSION;

    bufferT_offset = exclusiveStorageSize;
    exclusiveStorageSize += nBufferT * nCudaThreads;

    tmp1_offset = exclusiveStorageSize;
    exclusiveStorageSize += nTemp1 * nCudaThreads;

    tmp3_offset = exclusiveStorageSize;
    exclusiveStorageSize += nTemp3 * nCudaThreads;

    printf("exclusiveStorageSize:%lu\n", exclusiveStorageSize);

    for (uint32_t g = 0; g < nGroups; g++) {
        CHECKCUDAERR(hipStreamCreate(&streams[g]));
        CHECKCUDAERR(hipMalloc(&exclusiveStorage[g], exclusiveStorageSize * sizeof(uint64_t)));
//        CHECKCUDAERR(hipMalloc(&constPols_d, starkInfo.nConstants * (subDomainSize + nextStride) * sizeof(uint64_t)));
//        CHECKCUDAERR(hipMalloc(&x_d, subDomainSize * sizeof(uint64_t)));
//        CHECKCUDAERR(hipMalloc(&zi_d, subDomainSize * sizeof(uint64_t)));
//        CHECKCUDAERR(hipMalloc(&pols_d, total_offsets * sizeof(uint64_t)));
//        CHECKCUDAERR(hipMalloc(&xDivXSubXi_d, 2 * subDomainSize * FIELD_EXTENSION * sizeof(uint64_t)));
//
//        CHECKCUDAERR(hipMalloc(&gBufferT_, nBufferT * nCudaThreads * sizeof(uint64_t)));
//        CHECKCUDAERR(hipMalloc(&tmp1_d, nTemp1 * nCudaThreads * sizeof(uint64_t)));
//        CHECKCUDAERR(hipMalloc(&tmp3_d, nTemp3 * nCudaThreads * sizeof(uint64_t)));
    }
}

void CHelpersStepsPackGPU::cleanupGPU() {
    hipFree(sharedStorage);
    for (uint32_t g = 0; g < nGroups; g++) {
        CHECKCUDAERR(hipStreamDestroy(streams[g]));
        hipFree(exclusiveStorage[g]);
    }
}

void CHelpersStepsPackGPU::compare(StepsParams &params, uint64_t row) {

    for (uint64_t s = 1; s < 11; s++) {
        if (offsetsStagesGPU[s] != MAX_U64) {
            printf("write s:%lu\n", s);
            writeDataToFile("gpu.txt", (uint64_t *)params.pols +offsetsStages[s] + row*nColsStages[s], (subDomainSize + nextStride) *nColsStages[s]);
        }
    }

    assert(0);
}

void CHelpersStepsPackGPU::calculateExpressions(StarkInfo &starkInfo, StepsParams &params, ParserArgs &parserArgs, ParserParams &parserParams) {

    CHECKCUDAERR(hipSetDevice(0));

    prepareGPU(starkInfo, params, parserArgs, parserParams);
    calculateExpressionsRowsGPU(starkInfo, params, parserArgs, parserParams, 0, domainSize-nrowsPack * nCudaThreads*nGroups);
    cleanupGPU();
    calculateExpressionsRows(starkInfo, params, parserArgs, parserParams, domainSize-nrowsPack * nCudaThreads * nGroups, domainSize);
    //compare(params, 0);
}

void CHelpersStepsPackGPU::calculateExpressionsRowsGPU(StarkInfo &starkInfo, StepsParams &params, ParserArgs &parserArgs, ParserParams &parserParams,
    uint64_t rowIni, uint64_t rowEnd){

    if(rowEnd < rowIni || rowEnd > domainSize || (rowEnd -rowIni) % nrowsPack != 0) {
        zklog.info("Invalid range for rowIni " + to_string(rowIni) + " and rowEnd " + to_string(rowEnd));
        exitProcess();
    }

    CHelpersStepsPackGPU *cHelpersSteps_d;
    CHECKCUDAERR(hipMalloc((void **)&(cHelpersSteps_d), sizeof(CHelpersStepsPackGPU)));
    CHECKCUDAERR(hipMemcpy(cHelpersSteps_d, this, sizeof(CHelpersStepsPackGPU), hipMemcpyHostToDevice));

    for (uint64_t i = rowIni; i < rowEnd; i+= nrowsPack*nCudaThreads*nGroups) {
        printf("rows:%lu\n", i);
        for (uint32_t g=0; g<nGroups; g++) {
            hipStream_t stream = streams[g];
            uint64_t row = i + nrowsPack*nCudaThreads*g;
            TimerStart(Memcpy_H_to_D);
            loadData(starkInfo, params, row, g);
            TimerStopAndLog(Memcpy_H_to_D);

            TimerStart(EXP_Kernel);
            loadPolinomialsGPU<<<(nCudaThreads+15)/16,16,0,stream>>>(cHelpersSteps_d, starkInfo.nConstants, parserParams.stage, g);
            pack_kernel<<<(nCudaThreads+15)/16,16,0,stream>>>(cHelpersSteps_d, g);
            storePolinomialsGPU<<<(nCudaThreads+15)/16,16,0,stream>>>(cHelpersSteps_d, g);
            TimerStopAndLog(EXP_Kernel);

            TimerStart(Memcpy_D_to_H);
            storeData(starkInfo, params, row, g);
            TimerStopAndLog(Memcpy_D_to_H);
        }
    }

    for (uint32_t g = 0; g < nGroups; g++) {
        CHECKCUDAERR(hipStreamSynchronize(streams[g]));
    }

    hipFree(cHelpersSteps_d);
}

void CHelpersStepsPackGPU::loadData(StarkInfo &starkInfo, StepsParams &params, uint64_t row, uint32_t g) {

    ConstantPolsStarks *constPols = domainExtended ? params.pConstPols2ns : params.pConstPols;
    Polinomial &x = domainExtended ? params.x_2ns : params.x_n;

    uint64_t *constPols_d = exclusiveStorage[g] + constPols_offset;
    uint64_t *x_d = exclusiveStorage[g] + x_offset;
    uint64_t *zi_d = exclusiveStorage[g] + zi_offset;
    uint64_t *pols_d = exclusiveStorage[g] + pols_offset;
    uint64_t *xDivXSubXi_d = exclusiveStorage[g] + xDivXSubXi_offset;

    hipStream_t stream = streams[g];

    // TODO may overflow and cycle
    CHECKCUDAERR(hipMemcpyAsync(constPols_d, ((Goldilocks::Element *)constPols->address()) + row * starkInfo.nConstants, starkInfo.nConstants * (subDomainSize + nextStride) * sizeof(uint64_t), hipMemcpyHostToDevice, stream));
    CHECKCUDAERR(hipMemcpyAsync(x_d, x[row], subDomainSize * sizeof(uint64_t), hipMemcpyHostToDevice, stream));
    CHECKCUDAERR(hipMemcpyAsync(zi_d, params.zi[row], subDomainSize * sizeof(uint64_t), hipMemcpyHostToDevice, stream));

    for (uint64_t s = 1; s < 11; s++) {
        if (offsetsStagesGPU[s] != MAX_U64) {
            CHECKCUDAERR(hipMemcpyAsync(pols_d + offsetsStagesGPU[s], &params.pols[offsetsStages[s] + row*nColsStages[s]], (subDomainSize+nextStride) *nColsStages[s] * sizeof(uint64_t), hipMemcpyHostToDevice, stream));
        }
    }

    CHECKCUDAERR(hipMemcpyAsync(xDivXSubXi_d, params.xDivXSubXi[row], subDomainSize *FIELD_EXTENSION * sizeof(uint64_t), hipMemcpyHostToDevice, stream));
    CHECKCUDAERR(hipMemcpyAsync(xDivXSubXi_d + subDomainSize *FIELD_EXTENSION, params.xDivXSubXi[domainSize + row], subDomainSize *FIELD_EXTENSION * sizeof(uint64_t), hipMemcpyHostToDevice, stream));
}

void CHelpersStepsPackGPU::storeData(StarkInfo &starkInfo, StepsParams &params, uint64_t row, uint32_t g) {
    uint64_t *pols_d = exclusiveStorage[g] + pols_offset;
    hipStream_t stream = streams[g];
    for (uint64_t s = 1; s < 11; s++) {
        if (offsetsStagesGPU[s] != MAX_U64) {
            CHECKCUDAERR(hipMemcpyAsync(&params.pols[offsetsStages[s] + row*nColsStages[s]], pols_d + offsetsStagesGPU[s], subDomainSize *nColsStages[s] * sizeof(uint64_t), hipMemcpyDeviceToHost, stream));
        }
    }
}

__global__ void loadPolinomialsGPU(CHelpersStepsPackGPU *cHelpersSteps, uint64_t nConstants, uint64_t stage, uint32_t g) {

    uint64_t nCudaThreads = cHelpersSteps->nCudaThreads;

    uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= nCudaThreads) {
        return;
    }

    uint64_t nrowsPack = cHelpersSteps->nrowsPack;
    uint64_t nextStride = cHelpersSteps->nextStride;
    uint64_t subDomainSize = cHelpersSteps->subDomainSize;
    uint64_t nBufferT = cHelpersSteps->nBufferT;

    uint64_t *nColsStages = cHelpersSteps->sharedStorage + cHelpersSteps->nColsStages_offset;
    uint64_t *nColsStagesAcc = cHelpersSteps->sharedStorage + cHelpersSteps->nColsStagesAcc_offset;
    uint64_t *offsetsStages = cHelpersSteps->sharedStorage + cHelpersSteps->offsetsStages_offset;

    gl64_t *bufferT_ = (gl64_t *)cHelpersSteps->exclusiveStorage[g] + cHelpersSteps->bufferT_offset + idx * nBufferT;
    gl64_t *pols = (gl64_t *)cHelpersSteps->exclusiveStorage[g] + cHelpersSteps->pols_offset;
    gl64_t *constPols = (gl64_t *)cHelpersSteps->exclusiveStorage[g] + cHelpersSteps->constPols_offset;

    uint64_t row = idx*nrowsPack;
    uint64_t nStages = 3;
    uint64_t nextStrides[2] = {0, nextStride};

    for(uint64_t k = 0; k < nConstants; ++k) {
        for(uint64_t o = 0; o < 2; ++o) {
            for(uint64_t j = 0; j < nrowsPack; ++j) {
                uint64_t l = (row + j + nextStrides[o]);
                bufferT_[(nColsStagesAcc[5*o] + k)*nrowsPack + j] = constPols[l * nConstants + k];
            }
        }
    }

    // Load x and Zi
    for(uint64_t j = 0; j < nrowsPack; ++j) {
        bufferT_[nConstants*nrowsPack + j] = (cHelpersSteps->exclusiveStorage[g] + cHelpersSteps->x_offset)[row + j];
    }
    for(uint64_t j = 0; j < nrowsPack; ++j) {
        bufferT_[(nConstants + 1)*nrowsPack + j] = (cHelpersSteps->exclusiveStorage[g] + cHelpersSteps->zi_offset)[row + j];
    }

    for(uint64_t s = 1; s <= nStages; ++s) {
        for(uint64_t k = 0; k < nColsStages[s]; ++k) {
            for(uint64_t o = 0; o < 2; ++o) {
                for(uint64_t j = 0; j < nrowsPack; ++j) {
                    uint64_t l = (row + j + nextStrides[o]);
                    bufferT_[(nColsStagesAcc[5*o + s] + k)*nrowsPack + j] = pols[offsetsStages[s] + l * nColsStages[s] + k];
                }
            }
        }
    }

    if(stage == 5) {
        for(uint64_t k = 0; k < nColsStages[nStages + 1]; ++k) {
            for(uint64_t o = 0; o < 2; ++o) {
                for(uint64_t j = 0; j < nrowsPack; ++j) {
                    uint64_t l = (row + j + nextStrides[o]); // % domainSize;
                    bufferT_[(nColsStagesAcc[5*o + nStages + 1] + k)*nrowsPack + j] = pols[offsetsStages[nStages + 1] + l * nColsStages[nStages + 1] + k];
                }
            }
        }

       // Load xDivXSubXi & xDivXSubWXi
       for(uint64_t d = 0; d < 2; ++d) {
           for(uint64_t i = 0; i < FIELD_EXTENSION; ++i) {
               for(uint64_t j = 0; j < nrowsPack; ++j) {
                  bufferT_[(nColsStagesAcc[11] + FIELD_EXTENSION*d + i)*nrowsPack + j] = (cHelpersSteps->exclusiveStorage[g] + cHelpersSteps->xDivXSubXi_offset)[(d*subDomainSize + row + j) * FIELD_EXTENSION + i];
               }
           }
       }
    }
}


__global__ void storePolinomialsGPU(CHelpersStepsPackGPU *cHelpersSteps, uint32_t g) {
    uint64_t nCudaThreads = cHelpersSteps->nCudaThreads;

    uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= nCudaThreads) {
        return;
    }

    bool domainExtended = cHelpersSteps->domainExtended;
    uint64_t nrowsPack = cHelpersSteps->nrowsPack;
    uint64_t nBufferT = cHelpersSteps->nBufferT;

    uint64_t row = idx*nrowsPack;

    uint64_t *nColsStages = cHelpersSteps->sharedStorage + cHelpersSteps->nColsStages_offset;
    uint64_t *nColsStagesAcc = cHelpersSteps->sharedStorage + cHelpersSteps->nColsStagesAcc_offset;
    uint64_t *offsetsStages = cHelpersSteps->sharedStorage + cHelpersSteps->offsetsStages_offset;

    gl64_t *bufferT_ = (gl64_t *)cHelpersSteps->exclusiveStorage[g] + cHelpersSteps->bufferT_offset + idx * nBufferT;
    gl64_t *pols = (gl64_t *)cHelpersSteps->exclusiveStorage[g] + cHelpersSteps->pols_offset;

    if(domainExtended) {
        // Store either polinomial f or polinomial q
        for(uint64_t k = 0; k < nColsStages[10]; ++k) {
            gl64_t *buffT = &bufferT_[(nColsStagesAcc[10] + k)* nrowsPack];
            gl64_t::copy_pack(nrowsPack, &pols[offsetsStages[10] + k + row * nColsStages[10]], nColsStages[10], buffT);
        }
    } else {
        uint64_t nStages = 3;
        for(uint64_t s = 2; s <= nStages + 1; ++s) {
            for(uint64_t k = 0; k < nColsStages[s]; ++k) {
                gl64_t *buffT = &bufferT_[(nColsStagesAcc[s] + k)* nrowsPack];
                gl64_t::copy_pack(nrowsPack, &pols[offsetsStages[s] + k + row * nColsStages[s]], nColsStages[s], buffT);
            }
        }
    }
}

__global__ void pack_kernel(CHelpersStepsPackGPU *cHelpersSteps, uint32_t g)
{
    uint64_t nCudaThreads = cHelpersSteps->nCudaThreads;

    uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= nCudaThreads) {
        return;
    }

    uint64_t nrowsPack = cHelpersSteps->nrowsPack;
    uint64_t nOps = cHelpersSteps->nOps;
    uint64_t nArgs = cHelpersSteps->nArgs;
    uint64_t nBufferT = cHelpersSteps->nBufferT;
    uint64_t nTemp1 = cHelpersSteps->nTemp1;
    uint64_t nTemp3 = cHelpersSteps->nTemp3;

    uint64_t *nColsStagesAcc = cHelpersSteps->sharedStorage + cHelpersSteps->nColsStagesAcc_offset;
    uint64_t *ops = cHelpersSteps->sharedStorage + cHelpersSteps->ops_offset;
    uint64_t *args = cHelpersSteps->sharedStorage + cHelpersSteps->args_offset;
    gl64_t *challenges = (gl64_t *)cHelpersSteps->sharedStorage + cHelpersSteps->challenges_offset;
    gl64_t *challenges_ops = (gl64_t *)cHelpersSteps->sharedStorage + cHelpersSteps->challenges_ops_offset;
    gl64_t *numbers_ = (gl64_t *)cHelpersSteps->sharedStorage + cHelpersSteps->numbers_offset;
    gl64_t *publics = (gl64_t *)cHelpersSteps->sharedStorage + cHelpersSteps->publics_offset;
    gl64_t *evals = (gl64_t *)cHelpersSteps->sharedStorage + cHelpersSteps->evals_offset;

    gl64_t *bufferT_ = (gl64_t *)cHelpersSteps->exclusiveStorage[g] + cHelpersSteps->bufferT_offset + idx * nBufferT;
    gl64_t *tmp1 = (gl64_t *)cHelpersSteps->exclusiveStorage[g] + cHelpersSteps->tmp1_offset + nTemp1*idx;
    gl64_t *tmp3 = (gl64_t *)cHelpersSteps->exclusiveStorage[g] + cHelpersSteps->tmp3_offset + nTemp3*idx;

    uint64_t i_args = 0;

    for (uint64_t kk = 0; kk < nOps; ++kk) {
        switch (ops[kk]) {
            case 0: {
                // COPY commit1 to commit1
                gl64_t::copy_pack(nrowsPack, &bufferT_[(nColsStagesAcc[args[i_args]] + args[i_args + 1]) * nrowsPack], &bufferT_[(nColsStagesAcc[args[i_args + 2]] + args[i_args + 3]) * nrowsPack]);
                i_args += 4;
                break;
            }
            case 1: {
                // OPERATION WITH DEST: commit1 - SRC0: commit1 - SRC1: commit1
                gl64_t::op_pack(nrowsPack, args[i_args], &bufferT_[(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * nrowsPack], &bufferT_[(nColsStagesAcc[args[i_args + 3]] + args[i_args + 4]) * nrowsPack], &bufferT_[(nColsStagesAcc[args[i_args + 5]] + args[i_args + 6]) * nrowsPack]);
                i_args += 7;
                break;
            }
            case 2: {
                // OPERATION WITH DEST: commit1 - SRC0: commit1 - SRC1: tmp1
                gl64_t::op_pack(nrowsPack, args[i_args], &bufferT_[(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * nrowsPack], &bufferT_[(nColsStagesAcc[args[i_args + 3]] + args[i_args + 4]) * nrowsPack], &tmp1[args[i_args + 5] * nrowsPack]);
                i_args += 6;
                break;
            }
            case 3: {
                // OPERATION WITH DEST: commit1 - SRC0: commit1 - SRC1: public
                gl64_t::op_pack(nrowsPack, args[i_args], &bufferT_[(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * nrowsPack], &bufferT_[(nColsStagesAcc[args[i_args + 3]] + args[i_args + 4]) * nrowsPack], &publics[args[i_args + 5] * nrowsPack]);
                i_args += 6;
                break;
            }
            case 4: {
                // OPERATION WITH DEST: commit1 - SRC0: commit1 - SRC1: number
                gl64_t::op_pack(nrowsPack, args[i_args], &bufferT_[(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * nrowsPack], &bufferT_[(nColsStagesAcc[args[i_args + 3]] + args[i_args + 4]) * nrowsPack], &numbers_[args[i_args + 5]*nrowsPack]);
                i_args += 6;
                break;
            }
            case 5: {
                // COPY tmp1 to commit1
                gl64_t::copy_pack(nrowsPack, &bufferT_[(nColsStagesAcc[args[i_args]] + args[i_args + 1]) * nrowsPack], &tmp1[args[i_args + 2] * nrowsPack]);
                i_args += 3;
                break;
            }
            case 6: {
                // OPERATION WITH DEST: commit1 - SRC0: tmp1 - SRC1: tmp1
                gl64_t::op_pack(nrowsPack, args[i_args], &bufferT_[(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * nrowsPack], &tmp1[args[i_args + 3] * nrowsPack], &tmp1[args[i_args + 4] * nrowsPack]);
                i_args += 5;
                break;
            }
            case 7: {
                // OPERATION WITH DEST: commit1 - SRC0: tmp1 - SRC1: public
                gl64_t::op_pack(nrowsPack, args[i_args], &bufferT_[(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * nrowsPack], &tmp1[args[i_args + 3] * nrowsPack], &publics[args[i_args + 4] * nrowsPack]);
                i_args += 5;
                break;
            }
            case 8: {
                // OPERATION WITH DEST: commit1 - SRC0: tmp1 - SRC1: number
                gl64_t::op_pack(nrowsPack, args[i_args], &bufferT_[(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * nrowsPack], &tmp1[args[i_args + 3] * nrowsPack], &numbers_[args[i_args + 4]*nrowsPack]);
                i_args += 5;
                break;
            }
            case 9: {
                // COPY public to commit1
                gl64_t::copy_pack(nrowsPack, &bufferT_[(nColsStagesAcc[args[i_args]] + args[i_args + 1]) * nrowsPack], &publics[args[i_args + 2] * nrowsPack]);
                i_args += 3;
                break;
            }
            case 10: {
                // OPERATION WITH DEST: commit1 - SRC0: public - SRC1: public
                gl64_t::op_pack(nrowsPack, args[i_args], &bufferT_[(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * nrowsPack], &publics[args[i_args + 3] * nrowsPack], &publics[args[i_args + 4] * nrowsPack]);
                i_args += 5;
                break;
            }
            case 11: {
                // OPERATION WITH DEST: commit1 - SRC0: public - SRC1: number
                gl64_t::op_pack(nrowsPack, args[i_args], &bufferT_[(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * nrowsPack], &publics[args[i_args + 3] * nrowsPack], &numbers_[args[i_args + 4]*nrowsPack]);
                i_args += 5;
                break;
            }
            case 12: {
                // COPY number to commit1
                gl64_t::copy_pack(nrowsPack, &bufferT_[(nColsStagesAcc[args[i_args]] + args[i_args + 1]) * nrowsPack], &numbers_[args[i_args + 2]*nrowsPack]);
                i_args += 3;
                break;
            }
            case 13: {
                // OPERATION WITH DEST: commit1 - SRC0: number - SRC1: number
                gl64_t::op_pack(nrowsPack, args[i_args], &bufferT_[(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * nrowsPack], &numbers_[args[i_args + 3]*nrowsPack], &numbers_[args[i_args + 4]*nrowsPack]);
                i_args += 5;
                break;
            }
            case 14: {
                // COPY commit1 to tmp1
                gl64_t::copy_pack(nrowsPack, &tmp1[args[i_args] * nrowsPack], &bufferT_[(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * nrowsPack]);
                i_args += 3;
                break;
            }
            case 15: {
                // OPERATION WITH DEST: tmp1 - SRC0: commit1 - SRC1: commit1
                gl64_t::op_pack(nrowsPack, args[i_args], &tmp1[args[i_args + 1] * nrowsPack], &bufferT_[(nColsStagesAcc[args[i_args + 2]] + args[i_args + 3]) * nrowsPack], &bufferT_[(nColsStagesAcc[args[i_args + 4]] + args[i_args + 5]) * nrowsPack]);
                i_args += 6;
                break;
            }
            case 16: {
                // OPERATION WITH DEST: tmp1 - SRC0: commit1 - SRC1: tmp1
                gl64_t::op_pack(nrowsPack, args[i_args], &tmp1[args[i_args + 1] * nrowsPack], &bufferT_[(nColsStagesAcc[args[i_args + 2]] + args[i_args + 3]) * nrowsPack], &tmp1[args[i_args + 4] * nrowsPack]);
                i_args += 5;
                break;
            }
            case 17: {
                // OPERATION WITH DEST: tmp1 - SRC0: commit1 - SRC1: public
                gl64_t::op_pack(nrowsPack, args[i_args], &tmp1[args[i_args + 1] * nrowsPack], &bufferT_[(nColsStagesAcc[args[i_args + 2]] + args[i_args + 3]) * nrowsPack], &publics[args[i_args + 4] * nrowsPack]);
                i_args += 5;
                break;
            }
            case 18: {
                // OPERATION WITH DEST: tmp1 - SRC0: commit1 - SRC1: number
                gl64_t::op_pack(nrowsPack, args[i_args], &tmp1[args[i_args + 1] * nrowsPack], &bufferT_[(nColsStagesAcc[args[i_args + 2]] + args[i_args + 3]) * nrowsPack], &numbers_[args[i_args + 4]*nrowsPack]);
                i_args += 5;
                break;
            }
            case 19: {
                // COPY tmp1 to tmp1
                gl64_t::copy_pack(nrowsPack, &tmp1[args[i_args] * nrowsPack], &tmp1[args[i_args + 1] * nrowsPack]);
                i_args += 2;
                break;
            }
            case 20: {
                // OPERATION WITH DEST: tmp1 - SRC0: tmp1 - SRC1: tmp1
                gl64_t::op_pack(nrowsPack, args[i_args], &tmp1[args[i_args + 1] * nrowsPack], &tmp1[args[i_args + 2] * nrowsPack], &tmp1[args[i_args + 3] * nrowsPack]);
                i_args += 4;
                break;
            }
            case 21: {
                // OPERATION WITH DEST: tmp1 - SRC0: tmp1 - SRC1: public
                gl64_t::op_pack(nrowsPack, args[i_args], &tmp1[args[i_args + 1] * nrowsPack], &tmp1[args[i_args + 2] * nrowsPack], &publics[args[i_args + 3] * nrowsPack]);
                i_args += 4;
                break;
            }
            case 22: {
                // OPERATION WITH DEST: tmp1 - SRC0: tmp1 - SRC1: number
                gl64_t::op_pack(nrowsPack, args[i_args], &tmp1[args[i_args + 1] * nrowsPack], &tmp1[args[i_args + 2] * nrowsPack], &numbers_[args[i_args + 3]*nrowsPack]);
                i_args += 4;
                break;
            }
            case 23: {
                // COPY public to tmp1
                gl64_t::copy_pack(nrowsPack, &tmp1[args[i_args] * nrowsPack], &publics[args[i_args + 1] * nrowsPack]);
                i_args += 2;
                break;
            }
            case 24: {
                // OPERATION WITH DEST: tmp1 - SRC0: public - SRC1: public
                gl64_t::op_pack(nrowsPack, args[i_args], &tmp1[args[i_args + 1] * nrowsPack], &publics[args[i_args + 2] * nrowsPack], &publics[args[i_args + 3] * nrowsPack]);
                i_args += 4;
                break;
            }
            case 25: {
                // OPERATION WITH DEST: tmp1 - SRC0: public - SRC1: number
                gl64_t::op_pack(nrowsPack, args[i_args], &tmp1[args[i_args + 1] * nrowsPack], &publics[args[i_args + 2] * nrowsPack], &numbers_[args[i_args + 3]*nrowsPack]);
                i_args += 4;
                break;
            }
            case 26: {
                // COPY number to tmp1
                gl64_t::copy_pack(nrowsPack, &tmp1[args[i_args] * nrowsPack], &numbers_[args[i_args + 1]*nrowsPack]);
                i_args += 2;
                break;
            }
            case 27: {
                // OPERATION WITH DEST: tmp1 - SRC0: number - SRC1: number
                gl64_t::op_pack(nrowsPack, args[i_args], &tmp1[args[i_args + 1] * nrowsPack], &numbers_[args[i_args + 2]*nrowsPack], &numbers_[args[i_args + 3]*nrowsPack]);
                i_args += 4;
                break;
            }
            case 28: {
                // OPERATION WITH DEST: commit3 - SRC0: commit3 - SRC1: commit1
                Goldilocks3GPU::op_31_pack(nrowsPack, args[i_args], &bufferT_[(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * nrowsPack], &bufferT_[(nColsStagesAcc[args[i_args + 3]] + args[i_args + 4]) * nrowsPack], &bufferT_[(nColsStagesAcc[args[i_args + 5]] + args[i_args + 6]) * nrowsPack]);
                i_args += 7;
                break;
            }
            case 29: {
                // OPERATION WITH DEST: commit3 - SRC0: commit3 - SRC1: tmp1
                Goldilocks3GPU::op_31_pack(nrowsPack, args[i_args], &bufferT_[(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * nrowsPack], &bufferT_[(nColsStagesAcc[args[i_args + 3]] + args[i_args + 4]) * nrowsPack], &tmp1[args[i_args + 5] * nrowsPack]);
                i_args += 6;
                break;
            }
            case 30: {
                // OPERATION WITH DEST: commit3 - SRC0: commit3 - SRC1: public
                Goldilocks3GPU::op_31_pack(nrowsPack, args[i_args], &bufferT_[(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * nrowsPack], &bufferT_[(nColsStagesAcc[args[i_args + 3]] + args[i_args + 4]) * nrowsPack], &publics[args[i_args + 5] * nrowsPack]);
                i_args += 6;
                break;
            }
            case 31: {
                // OPERATION WITH DEST: commit3 - SRC0: commit3 - SRC1: number
                Goldilocks3GPU::op_31_pack(nrowsPack, args[i_args], &bufferT_[(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * nrowsPack], &bufferT_[(nColsStagesAcc[args[i_args + 3]] + args[i_args + 4]) * nrowsPack], &numbers_[args[i_args + 5]*nrowsPack]);
                i_args += 6;
                break;
            }
            case 32: {
                // OPERATION WITH DEST: commit3 - SRC0: tmp3 - SRC1: commit1
                Goldilocks3GPU::op_31_pack(nrowsPack, args[i_args], &bufferT_[(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * nrowsPack], &tmp3[args[i_args + 3] * nrowsPack * FIELD_EXTENSION], &bufferT_[(nColsStagesAcc[args[i_args + 4]] + args[i_args + 5]) * nrowsPack]);
                i_args += 6;
                break;
            }
            case 33: {
                // OPERATION WITH DEST: commit3 - SRC0: tmp3 - SRC1: tmp1
                Goldilocks3GPU::op_31_pack(nrowsPack, args[i_args], &bufferT_[(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * nrowsPack], &tmp3[args[i_args + 3] * nrowsPack * FIELD_EXTENSION], &tmp1[args[i_args + 4] * nrowsPack]);
                i_args += 5;
                break;
            }
            case 34: {
                // OPERATION WITH DEST: commit3 - SRC0: tmp3 - SRC1: public
                Goldilocks3GPU::op_31_pack(nrowsPack, args[i_args], &bufferT_[(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * nrowsPack], &tmp3[args[i_args + 3] * nrowsPack * FIELD_EXTENSION], &publics[args[i_args + 4] * nrowsPack]);
                i_args += 5;
                break;
            }
            case 35: {
                // OPERATION WITH DEST: commit3 - SRC0: tmp3 - SRC1: number
                Goldilocks3GPU::op_31_pack(nrowsPack, args[i_args], &bufferT_[(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * nrowsPack], &tmp3[args[i_args + 3] * nrowsPack * FIELD_EXTENSION], &numbers_[args[i_args + 4]*nrowsPack]);
                i_args += 5;
                break;
            }
            case 36: {
                // OPERATION WITH DEST: commit3 - SRC0: challenge - SRC1: commit1
                Goldilocks3GPU::op_31_pack(nrowsPack, args[i_args], &bufferT_[(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * nrowsPack], &challenges[args[i_args + 3]*FIELD_EXTENSION*nrowsPack], &bufferT_[(nColsStagesAcc[args[i_args + 4]] + args[i_args + 5]) * nrowsPack]);
                i_args += 6;
                break;
            }
            case 37: {
                // OPERATION WITH DEST: commit3 - SRC0: challenge - SRC1: tmp1
                Goldilocks3GPU::op_31_pack(nrowsPack, args[i_args], &bufferT_[(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * nrowsPack], &challenges[args[i_args + 3]*FIELD_EXTENSION*nrowsPack], &tmp1[args[i_args + 4] * nrowsPack]);
                i_args += 5;
                break;
            }
            case 38: {
                // OPERATION WITH DEST: commit3 - SRC0: challenge - SRC1: public
                Goldilocks3GPU::op_31_pack(nrowsPack, args[i_args], &bufferT_[(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * nrowsPack], &challenges[args[i_args + 3]*FIELD_EXTENSION*nrowsPack], &publics[args[i_args + 4] * nrowsPack]);
                i_args += 5;
                break;
            }
            case 39: {
                // OPERATION WITH DEST: commit3 - SRC0: challenge - SRC1: number
                Goldilocks3GPU::op_31_pack(nrowsPack, args[i_args], &bufferT_[(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * nrowsPack], &challenges[args[i_args + 3]*FIELD_EXTENSION*nrowsPack], &numbers_[args[i_args + 4]*nrowsPack]);
                i_args += 5;
                break;
            }
            case 40: {
                // COPY commit3 to commit3
                Goldilocks3GPU::copy_pack(nrowsPack, &bufferT_[(nColsStagesAcc[args[i_args]] + args[i_args + 1]) * nrowsPack], &bufferT_[(nColsStagesAcc[args[i_args + 2]] + args[i_args + 3]) * nrowsPack]);
                i_args += 4;
                break;
            }
            case 41: {
                // OPERATION WITH DEST: commit3 - SRC0: commit3 - SRC1: commit3
                Goldilocks3GPU::op_pack(nrowsPack, args[i_args], &bufferT_[(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * nrowsPack], &bufferT_[(nColsStagesAcc[args[i_args + 3]] + args[i_args + 4]) * nrowsPack], &bufferT_[(nColsStagesAcc[args[i_args + 5]] + args[i_args + 6]) * nrowsPack]);
                i_args += 7;
                break;
            }
            case 42: {
                // OPERATION WITH DEST: commit3 - SRC0: commit3 - SRC1: tmp3
                Goldilocks3GPU::op_pack(nrowsPack, args[i_args], &bufferT_[(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * nrowsPack], &bufferT_[(nColsStagesAcc[args[i_args + 3]] + args[i_args + 4]) * nrowsPack], &tmp3[args[i_args + 5] * nrowsPack * FIELD_EXTENSION]);
                i_args += 6;
                break;
            }
            case 43: {
                // MULTIPLICATION WITH DEST: commit3 - SRC0: commit3 - SRC1: challenge
                Goldilocks3GPU::mul_pack(nrowsPack, &bufferT_[(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * nrowsPack], &bufferT_[(nColsStagesAcc[args[i_args + 3]] + args[i_args + 4]) * nrowsPack], &challenges[args[i_args + 5]*FIELD_EXTENSION*nrowsPack], &challenges_ops[args[i_args + 5]*FIELD_EXTENSION*nrowsPack]);
                i_args += 6;
                break;
            }
            case 44: {
                // OPERATION WITH DEST: commit3 - SRC0: commit3 - SRC1: challenge
                Goldilocks3GPU::op_pack(nrowsPack, args[i_args], &bufferT_[(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * nrowsPack], &bufferT_[(nColsStagesAcc[args[i_args + 3]] + args[i_args + 4]) * nrowsPack], &challenges[args[i_args + 5]*FIELD_EXTENSION*nrowsPack]);
                i_args += 6;
                break;
            }
            case 45: {
                // COPY tmp3 to commit3
                Goldilocks3GPU::copy_pack(nrowsPack, &bufferT_[(nColsStagesAcc[args[i_args]] + args[i_args + 1]) * nrowsPack], &tmp3[args[i_args + 2] * nrowsPack * FIELD_EXTENSION]);
                i_args += 3;
                break;
            }
            case 46: {
                // OPERATION WITH DEST: commit3 - SRC0: tmp3 - SRC1: tmp3
                Goldilocks3GPU::op_pack(nrowsPack, args[i_args], &bufferT_[(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * nrowsPack], &tmp3[args[i_args + 3] * nrowsPack * FIELD_EXTENSION], &tmp3[args[i_args + 4] * nrowsPack * FIELD_EXTENSION]);
                i_args += 5;
                break;
            }
            case 47: {
                // MULTIPLICATION WITH DEST: commit3 - SRC0: tmp3 - SRC1: challenge
                Goldilocks3GPU::mul_pack(nrowsPack, &bufferT_[(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * nrowsPack], &tmp3[args[i_args + 3] * nrowsPack * FIELD_EXTENSION], &challenges[args[i_args + 4]*FIELD_EXTENSION*nrowsPack], &challenges_ops[args[i_args + 4]*FIELD_EXTENSION*nrowsPack]);
                i_args += 5;
                break;
            }
            case 48: {
                // OPERATION WITH DEST: commit3 - SRC0: tmp3 - SRC1: challenge
                Goldilocks3GPU::op_pack(nrowsPack, args[i_args], &bufferT_[(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * nrowsPack], &tmp3[args[i_args + 3] * nrowsPack * FIELD_EXTENSION], &challenges[args[i_args + 4]*FIELD_EXTENSION*nrowsPack]);
                i_args += 5;
                break;
            }
            case 49: {
                // MULTIPLICATION WITH DEST: commit3 - SRC0: challenge - SRC1: challenge
                Goldilocks3GPU::mul_pack(nrowsPack, &bufferT_[(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * nrowsPack], &challenges[args[i_args + 3]*FIELD_EXTENSION*nrowsPack], &challenges[args[i_args + 4]*FIELD_EXTENSION*nrowsPack], &challenges_ops[args[i_args + 4]*FIELD_EXTENSION*nrowsPack]);
                i_args += 5;
                break;
            }
            case 50: {
                // OPERATION WITH DEST: commit3 - SRC0: challenge - SRC1: challenge
                Goldilocks3GPU::op_pack(nrowsPack, args[i_args], &bufferT_[(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * nrowsPack], &challenges[args[i_args + 3]*FIELD_EXTENSION*nrowsPack], &challenges[args[i_args + 4]*FIELD_EXTENSION*nrowsPack]);
                i_args += 5;
                break;
            }
            case 51: {
                // OPERATION WITH DEST: tmp3 - SRC0: commit3 - SRC1: commit1
                Goldilocks3GPU::op_31_pack(nrowsPack, args[i_args], &tmp3[args[i_args + 1] * nrowsPack * FIELD_EXTENSION], &bufferT_[(nColsStagesAcc[args[i_args + 2]] + args[i_args + 3]) * nrowsPack], &bufferT_[(nColsStagesAcc[args[i_args + 4]] + args[i_args + 5]) * nrowsPack]);
                i_args += 6;
                break;
            }
            case 52: {
                // OPERATION WITH DEST: tmp3 - SRC0: commit3 - SRC1: tmp1
                Goldilocks3GPU::op_31_pack(nrowsPack, args[i_args], &tmp3[args[i_args + 1] * nrowsPack * FIELD_EXTENSION], &bufferT_[(nColsStagesAcc[args[i_args + 2]] + args[i_args + 3]) * nrowsPack], &tmp1[args[i_args + 4] * nrowsPack]);
                i_args += 5;
                break;
            }
            case 53: {
                // OPERATION WITH DEST: tmp3 - SRC0: commit3 - SRC1: public
                Goldilocks3GPU::op_31_pack(nrowsPack, args[i_args], &tmp3[args[i_args + 1] * nrowsPack * FIELD_EXTENSION], &bufferT_[(nColsStagesAcc[args[i_args + 2]] + args[i_args + 3]) * nrowsPack], &publics[args[i_args + 4] * nrowsPack]);
                i_args += 5;
                break;
            }
            case 54: {
                // OPERATION WITH DEST: tmp3 - SRC0: commit3 - SRC1: number
                Goldilocks3GPU::op_31_pack(nrowsPack, args[i_args], &tmp3[args[i_args + 1] * nrowsPack * FIELD_EXTENSION], &bufferT_[(nColsStagesAcc[args[i_args + 2]] + args[i_args + 3]) * nrowsPack], &numbers_[args[i_args + 4]*nrowsPack]);
                i_args += 5;
                break;
            }
            case 55: {
                // OPERATION WITH DEST: tmp3 - SRC0: tmp3 - SRC1: commit1
                Goldilocks3GPU::op_31_pack(nrowsPack, args[i_args], &tmp3[args[i_args + 1] * nrowsPack * FIELD_EXTENSION], &tmp3[args[i_args + 2] * nrowsPack * FIELD_EXTENSION], &bufferT_[(nColsStagesAcc[args[i_args + 3]] + args[i_args + 4]) * nrowsPack]);
                i_args += 5;
                break;
            }
            case 56: {
                // OPERATION WITH DEST: tmp3 - SRC0: tmp3 - SRC1: tmp1
                Goldilocks3GPU::op_31_pack(nrowsPack, args[i_args], &tmp3[args[i_args + 1] * nrowsPack * FIELD_EXTENSION], &tmp3[args[i_args + 2] * nrowsPack * FIELD_EXTENSION], &tmp1[args[i_args + 3] * nrowsPack]);
                i_args += 4;
                break;
            }
            case 57: {
                // OPERATION WITH DEST: tmp3 - SRC0: tmp3 - SRC1: public
                Goldilocks3GPU::op_31_pack(nrowsPack, args[i_args], &tmp3[args[i_args + 1] * nrowsPack * FIELD_EXTENSION], &tmp3[args[i_args + 2] * nrowsPack * FIELD_EXTENSION], &publics[args[i_args + 3] * nrowsPack]);
                i_args += 4;
                break;
            }
            case 58: {
                // OPERATION WITH DEST: tmp3 - SRC0: tmp3 - SRC1: number
                Goldilocks3GPU::op_31_pack(nrowsPack, args[i_args], &tmp3[args[i_args + 1] * nrowsPack * FIELD_EXTENSION], &tmp3[args[i_args + 2] * nrowsPack * FIELD_EXTENSION], &numbers_[args[i_args + 3]*nrowsPack]);
                i_args += 4;
                break;
            }
            case 59: {
                // OPERATION WITH DEST: tmp3 - SRC0: challenge - SRC1: commit1
                Goldilocks3GPU::op_31_pack(nrowsPack, args[i_args], &tmp3[args[i_args + 1] * nrowsPack * FIELD_EXTENSION], &challenges[args[i_args + 2]*FIELD_EXTENSION*nrowsPack], &bufferT_[(nColsStagesAcc[args[i_args + 3]] + args[i_args + 4]) * nrowsPack]);
                i_args += 5;
                break;
            }
            case 60: {
                // OPERATION WITH DEST: tmp3 - SRC0: challenge - SRC1: tmp1
                Goldilocks3GPU::op_31_pack(nrowsPack, args[i_args], &tmp3[args[i_args + 1] * nrowsPack * FIELD_EXTENSION], &challenges[args[i_args + 2]*FIELD_EXTENSION*nrowsPack], &tmp1[args[i_args + 3] * nrowsPack]);
                i_args += 4;
                break;
            }
            case 61: {
                // OPERATION WITH DEST: tmp3 - SRC0: challenge - SRC1: public
                Goldilocks3GPU::op_31_pack(nrowsPack, args[i_args], &tmp3[args[i_args + 1] * nrowsPack * FIELD_EXTENSION], &challenges[args[i_args + 2]*FIELD_EXTENSION*nrowsPack], &publics[args[i_args + 3] * nrowsPack]);
                i_args += 4;
                break;
            }
            case 62: {
                // OPERATION WITH DEST: tmp3 - SRC0: challenge - SRC1: number
                Goldilocks3GPU::op_31_pack(nrowsPack, args[i_args], &tmp3[args[i_args + 1] * nrowsPack * FIELD_EXTENSION], &challenges[args[i_args + 2]*FIELD_EXTENSION*nrowsPack], &numbers_[args[i_args + 3]*nrowsPack]);
                i_args += 4;
                break;
            }
            case 63: {
                // COPY commit3 to tmp3
                Goldilocks3GPU::copy_pack(nrowsPack, &tmp3[args[i_args] * nrowsPack * FIELD_EXTENSION], &bufferT_[(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * nrowsPack]);
                i_args += 3;
                break;
            }
            case 64: {
                // OPERATION WITH DEST: tmp3 - SRC0: commit3 - SRC1: commit3
                Goldilocks3GPU::op_pack(nrowsPack, args[i_args], &tmp3[args[i_args + 1] * nrowsPack * FIELD_EXTENSION], &bufferT_[(nColsStagesAcc[args[i_args + 2]] + args[i_args + 3]) * nrowsPack], &bufferT_[(nColsStagesAcc[args[i_args + 4]] + args[i_args + 5]) * nrowsPack]);
                i_args += 6;
                break;
            }
            case 65: {
                // OPERATION WITH DEST: tmp3 - SRC0: commit3 - SRC1: tmp3
                Goldilocks3GPU::op_pack(nrowsPack, args[i_args], &tmp3[args[i_args + 1] * nrowsPack * FIELD_EXTENSION], &bufferT_[(nColsStagesAcc[args[i_args + 2]] + args[i_args + 3]) * nrowsPack], &tmp3[args[i_args + 4] * nrowsPack * FIELD_EXTENSION]);
                i_args += 5;
                break;
            }
            case 66: {
                // MULTIPLICATION WITH DEST: tmp3 - SRC0: commit3 - SRC1: challenge
                Goldilocks3GPU::mul_pack(nrowsPack, &tmp3[args[i_args + 1] * nrowsPack * FIELD_EXTENSION], &bufferT_[(nColsStagesAcc[args[i_args + 2]] + args[i_args + 3]) * nrowsPack], &challenges[args[i_args + 4]*FIELD_EXTENSION*nrowsPack], &challenges_ops[args[i_args + 4]*FIELD_EXTENSION*nrowsPack]);
                i_args += 5;
                break;
            }
            case 67: {
                // OPERATION WITH DEST: tmp3 - SRC0: commit3 - SRC1: challenge
                Goldilocks3GPU::op_pack(nrowsPack, args[i_args], &tmp3[args[i_args + 1] * nrowsPack * FIELD_EXTENSION], &bufferT_[(nColsStagesAcc[args[i_args + 2]] + args[i_args + 3]) * nrowsPack], &challenges[args[i_args + 4]*FIELD_EXTENSION*nrowsPack]);
                i_args += 5;
                break;
            }
            case 68: {
                // COPY tmp3 to tmp3
                Goldilocks3GPU::copy_pack(nrowsPack, &tmp3[args[i_args] * nrowsPack * FIELD_EXTENSION], &tmp3[args[i_args + 1] * nrowsPack * FIELD_EXTENSION]);
                i_args += 2;
                break;
            }
            case 69: {
                // OPERATION WITH DEST: tmp3 - SRC0: tmp3 - SRC1: tmp3
                Goldilocks3GPU::op_pack(nrowsPack, args[i_args], &tmp3[args[i_args + 1] * nrowsPack * FIELD_EXTENSION], &tmp3[args[i_args + 2] * nrowsPack * FIELD_EXTENSION], &tmp3[args[i_args + 3] * nrowsPack * FIELD_EXTENSION]);
                i_args += 4;
                break;
            }
            case 70: {
                // MULTIPLICATION WITH DEST: tmp3 - SRC0: tmp3 - SRC1: challenge
                Goldilocks3GPU::mul_pack(nrowsPack, &tmp3[args[i_args + 1] * nrowsPack * FIELD_EXTENSION], &tmp3[args[i_args + 2] * nrowsPack * FIELD_EXTENSION], &challenges[args[i_args + 3]*FIELD_EXTENSION*nrowsPack], &challenges_ops[args[i_args + 3]*FIELD_EXTENSION*nrowsPack]);
                i_args += 4;
                break;
            }
            case 71: {
                // OPERATION WITH DEST: tmp3 - SRC0: tmp3 - SRC1: challenge
                Goldilocks3GPU::op_pack(nrowsPack, args[i_args], &tmp3[args[i_args + 1] * nrowsPack * FIELD_EXTENSION], &tmp3[args[i_args + 2] * nrowsPack * FIELD_EXTENSION], &challenges[args[i_args + 3]*FIELD_EXTENSION*nrowsPack]);
                i_args += 4;
                break;
            }
            case 72: {
                // MULTIPLICATION WITH DEST: tmp3 - SRC0: challenge - SRC1: challenge
                Goldilocks3GPU::mul_pack(nrowsPack, &tmp3[args[i_args + 1] * nrowsPack * FIELD_EXTENSION], &challenges[args[i_args + 2]*FIELD_EXTENSION*nrowsPack], &challenges[args[i_args + 3]*FIELD_EXTENSION*nrowsPack], &challenges_ops[args[i_args + 3]*FIELD_EXTENSION*nrowsPack]);
                i_args += 4;
                break;
            }
            case 73: {
                // OPERATION WITH DEST: tmp3 - SRC0: challenge - SRC1: challenge
                Goldilocks3GPU::op_pack(nrowsPack, args[i_args], &tmp3[args[i_args + 1] * nrowsPack * FIELD_EXTENSION], &challenges[args[i_args + 2]*FIELD_EXTENSION*nrowsPack], &challenges[args[i_args + 3]*FIELD_EXTENSION*nrowsPack]);
                i_args += 4;
                break;
            }
            case 74: {
                // COPY eval to tmp3
                Goldilocks3GPU::copy_pack(nrowsPack, &tmp3[args[i_args] * nrowsPack * FIELD_EXTENSION], &evals[args[i_args + 1]*FIELD_EXTENSION*nrowsPack]);
                i_args += 2;
                break;
            }
            case 75: {
                // MULTIPLICATION WITH DEST: tmp3 - SRC0: eval - SRC1: challenge
                Goldilocks3GPU::mul_pack(nrowsPack, &tmp3[args[i_args + 1] * nrowsPack * FIELD_EXTENSION], &evals[args[i_args + 2]*FIELD_EXTENSION*nrowsPack], &challenges[args[i_args + 3]*FIELD_EXTENSION*nrowsPack], &challenges_ops[args[i_args + 3]*FIELD_EXTENSION*nrowsPack]);
                i_args += 4;
                break;
            }
            case 76: {
                // OPERATION WITH DEST: tmp3 - SRC0: challenge - SRC1: eval
                Goldilocks3GPU::op_pack(nrowsPack, args[i_args], &tmp3[args[i_args + 1] * nrowsPack * FIELD_EXTENSION], &challenges[args[i_args + 2]*FIELD_EXTENSION*nrowsPack], &evals[args[i_args + 3]*FIELD_EXTENSION*nrowsPack]);
                i_args += 4;
                break;
            }
            case 77: {
                // OPERATION WITH DEST: tmp3 - SRC0: tmp3 - SRC1: eval
                Goldilocks3GPU::op_pack(nrowsPack, args[i_args], &tmp3[args[i_args + 1] * nrowsPack * FIELD_EXTENSION], &tmp3[args[i_args + 2] * nrowsPack * FIELD_EXTENSION], &evals[args[i_args + 3]*FIELD_EXTENSION*nrowsPack]);
                i_args += 4;
                break;
            }
            case 78: {
                // OPERATION WITH DEST: tmp3 - SRC0: eval - SRC1: commit1
                Goldilocks3GPU::op_31_pack(nrowsPack, args[i_args], &tmp3[args[i_args + 1] * nrowsPack * FIELD_EXTENSION], &evals[args[i_args + 2]*FIELD_EXTENSION*nrowsPack], &bufferT_[(nColsStagesAcc[args[i_args + 3]] + args[i_args + 4]) * nrowsPack]);
                i_args += 5;
                break;
            }
            case 79: {
                // OPERATION WITH DEST: tmp3 - SRC0: commit3 - SRC1: eval
                Goldilocks3GPU::op_pack(nrowsPack, args[i_args], &tmp3[args[i_args + 1] * nrowsPack * FIELD_EXTENSION], &bufferT_[(nColsStagesAcc[args[i_args + 2]] + args[i_args + 3]) * nrowsPack], &evals[args[i_args + 4]*FIELD_EXTENSION*nrowsPack]);
                i_args += 5;
                break;
            }
            default: {
                assert(false);
            }
        }
    }

    assert(i_args == nArgs);

}

#endif