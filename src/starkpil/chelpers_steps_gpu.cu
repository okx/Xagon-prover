#include "hip/hip_runtime.h"
#include "chelpers_steps_gpu.hpp"
#include "chelpers_steps_pack.hpp"
#include "zklog.hpp"

#if defined(__USE_CUDA__) && defined(ENABLE_EXPERIMENTAL_CODE) && defined(YXQ)

#include "gl64_t.cuh"
#include "goldilocks_cubic_extension.cuh"
#include <inttypes.h>
#include "cuda_utils.cuh"
#include "cuda_utils.hpp"

void CHelpersStepsGPU::dataSetup(StarkInfo &starkInfo, StepsParams &params, ParserArgs &parserArgs, ParserParams &parserParams)
{
    bool domainExtended = parserParams.stage <= 3 ? false : true;
    uint64_t domainSize = domainExtended ? 1 << starkInfo.starkStruct.nBitsExt : 1 << starkInfo.starkStruct.nBits;
    uint64_t nextStride = domainExtended ? 1 << (starkInfo.starkStruct.nBitsExt - starkInfo.starkStruct.nBits) : 1;

    /*
        Metadata
    */
    nColsStagesAcc.resize(10 + 2);
    nColsStages.resize(10 + 2);
    offsetsStages.resize(10 + 2);

    nColsStages[0] = starkInfo.nConstants + 2;
    offsetsStages[0] = 0;

    for (uint64_t s = 1; s <= 3; ++s)
    {
        nColsStages[s] = starkInfo.mapSectionsN.section[string2section("cm" + to_string(s) + "_n")];
        if (domainExtended)
        {
            offsetsStages[s] = starkInfo.mapOffsets.section[string2section("cm" + to_string(s) + "_2ns")];
        }
        else
        {
            offsetsStages[s] = starkInfo.mapOffsets.section[string2section("cm" + to_string(s) + "_n")];
        }
    }
    if (domainExtended)
    {
        nColsStages[4] = starkInfo.mapSectionsN.section[eSection::cm4_2ns];
        offsetsStages[4] = starkInfo.mapOffsets.section[eSection::cm4_2ns];
    }
    else
    {
        nColsStages[4] = starkInfo.mapSectionsN.section[eSection::tmpExp_n];
        offsetsStages[4] = starkInfo.mapOffsets.section[eSection::tmpExp_n];
    }
    for (uint64_t o = 0; o < 2; ++o)
    {
        for (uint64_t s = 0; s < 5; ++s)
        {
            if (s == 0)
            {
                if (o == 0)
                {
                    nColsStagesAcc[0] = 0;
                }
                else
                {
                    nColsStagesAcc[5 * o] = nColsStagesAcc[5 * o - 1] + nColsStages[4];
                }
            }
            else
            {
                nColsStagesAcc[5 * o + s] = nColsStagesAcc[5 * o + (s - 1)] + nColsStages[(s - 1)];
            }
        }
    }
    nColsStagesAcc[10] = nColsStagesAcc[9] + nColsStages[4]; // Polinomials f & q
    if (parserParams.stage == 4)
    {
        offsetsStages[10] = starkInfo.mapOffsets.section[eSection::q_2ns];
        nColsStages[10] = starkInfo.qDim;
    }
    else if (parserParams.stage == 5)
    {
        offsetsStages[10] = starkInfo.mapOffsets.section[eSection::f_2ns];
        nColsStages[10] = 3;
    }
    nColsStagesAcc[11] = nColsStagesAcc[10] + nColsStages[10]; // xDivXSubXi
    nCols = nColsStagesAcc[11] + 6; // 3 for xDivXSubXi and 3 for xDivXSubWxi
    
    stepPointers_h.domainSize = domainSize;
    stepPointers_h.nConstants = starkInfo.nConstants;
    stepPointers_h.nextStride = nextStride;
    
    CHECKCUDAERR(hipMalloc((void **)&(stepPointers_h.nColsStages_d), nColsStages.size() * sizeof(uint64_t)));
    CHECKCUDAERR(hipMemcpy(stepPointers_h.nColsStages_d, nColsStages.data(), nColsStages.size() * sizeof(uint64_t), hipMemcpyHostToDevice));

    CHECKCUDAERR(hipMalloc((void **)&(stepPointers_h.nColsStagesAcc_d), nColsStagesAcc.size() * sizeof(uint64_t)));
    CHECKCUDAERR(hipMemcpy(stepPointers_h.nColsStagesAcc_d, nColsStagesAcc.data(), nColsStagesAcc.size() * sizeof(uint64_t), hipMemcpyHostToDevice));

    /*
        non-buffered data
    */
    uint8_t *ops = &parserArgs.ops[parserParams.opsOffset];
    uint32_t *ops_aux = new uint32_t[parserParams.nOps];
    for (uint64_t i = 0; i < parserParams.nOps; ++i)
        ops_aux[i] = uint32_t(ops[i]);
    CHECKCUDAERR(hipMalloc((void **)&(stepPointers_h.ops_d), parserParams.nOps * sizeof(uint32_t)));
    CHECKCUDAERR(hipMemcpy(stepPointers_h.ops_d, ops_aux, parserParams.nOps * sizeof(uint32_t), hipMemcpyHostToDevice));
    delete[] ops_aux;

    uint16_t *args = &parserArgs.args[parserParams.argsOffset];
    uint32_t *args_aux = new uint32_t[parserParams.nArgs];
    for (uint64_t i = 0; i < parserParams.nArgs; ++i)
        args_aux[i] = uint32_t(args[i]);
    CHECKCUDAERR(hipMalloc((void **)&(stepPointers_h.args_d), parserParams.nArgs * sizeof(uint32_t)));
    CHECKCUDAERR(hipMemcpy(stepPointers_h.args_d, args_aux, parserParams.nArgs * sizeof(uint32_t), hipMemcpyHostToDevice));
    delete[] args_aux;

    uint64_t *numbers = &parserArgs.numbers[parserParams.numbersOffset];
    Goldilocks::Element *numbers_aux = new Goldilocks::Element[parserParams.nNumbers * nrowsPack];
    //this expansion could be done in the GPU...
    for (uint64_t i = 0; i < parserParams.nNumbers; ++i)
    {
        for (uint64_t j = 0; j < nrowsPack; ++j)
        {
            numbers_aux[i * nrowsPack + j] = Goldilocks::fromU64(numbers[i]);
        }
    }
    CHECKCUDAERR(hipMalloc((void **)&(stepPointers_h.numbers_d), parserParams.nNumbers * nrowsPack * sizeof(gl64_t)));
    CHECKCUDAERR(hipMemcpy(stepPointers_h.numbers_d, numbers_aux, parserParams.nNumbers * nrowsPack * sizeof(gl64_t), hipMemcpyHostToDevice));
    delete[] numbers_aux;

    Goldilocks::Element *challenges_aux = new Goldilocks::Element[params.challenges.degree() * FIELD_EXTENSION * nrowsPack];
    Goldilocks::Element *challenges_ops_aux = new Goldilocks::Element[params.challenges.degree() * FIELD_EXTENSION * nrowsPack];
    //this expansion could be done in the GPU...
    for (uint64_t i = 0; i < params.challenges.degree(); ++i)
    {
        for (uint64_t j = 0; j < nrowsPack; ++j)
        {
            challenges_aux[(i * FIELD_EXTENSION) * nrowsPack + j] = params.challenges[i][0];
            challenges_aux[(i * FIELD_EXTENSION + 1) * nrowsPack + j] = params.challenges[i][1];
            challenges_aux[(i * FIELD_EXTENSION + 2) * nrowsPack + j] = params.challenges[i][2];
            challenges_ops_aux[(i * FIELD_EXTENSION) * nrowsPack + j] = params.challenges[i][0] + params.challenges[i][1];
            challenges_ops_aux[(i * FIELD_EXTENSION + 1) * nrowsPack + j] = params.challenges[i][0] + params.challenges[i][2];
            challenges_ops_aux[(i * FIELD_EXTENSION + 2) * nrowsPack + j] = params.challenges[i][1] + params.challenges[i][2];
        }
    }
    

    CHECKCUDAERR(hipMalloc((void **)&(stepPointers_h.challenges_d), params.challenges.degree() * FIELD_EXTENSION * nrowsPack * sizeof(gl64_t)));
    CHECKCUDAERR(hipMemcpy(stepPointers_h.challenges_d, challenges_aux, params.challenges.degree() * FIELD_EXTENSION * nrowsPack * sizeof(gl64_t), hipMemcpyHostToDevice));

    CHECKCUDAERR(hipMalloc((void **)&(stepPointers_h.challenges_ops_d), params.challenges.degree() * FIELD_EXTENSION * nrowsPack * sizeof(gl64_t)));
    CHECKCUDAERR(hipMemcpy(stepPointers_h.challenges_ops_d, challenges_ops_aux, params.challenges.degree() * FIELD_EXTENSION * nrowsPack * sizeof(gl64_t), hipMemcpyHostToDevice));

    delete[] challenges_aux;
    delete[] challenges_ops_aux;

    Goldilocks::Element *publics_aux = new Goldilocks::Element[starkInfo.nPublics * nrowsPack];
    for (uint64_t i = 0; i < starkInfo.nPublics; ++i)
    {
        for (uint64_t j = 0; j < nrowsPack; ++j)
        {
            publics_aux[i * nrowsPack + j] = params.publicInputs[i];
        }
    }
    CHECKCUDAERR(hipMalloc((void **)&(stepPointers_h.publics_d), starkInfo.nPublics * nrowsPack * sizeof(gl64_t)));
    CHECKCUDAERR(hipMemcpy(stepPointers_h.publics_d, publics_aux, starkInfo.nPublics * nrowsPack * sizeof(gl64_t), hipMemcpyHostToDevice));
    delete[] publics_aux;

    Goldilocks::Element *evals_aux = new Goldilocks::Element[params.evals.degree() * FIELD_EXTENSION * nrowsPack];
    for (uint64_t i = 0; i < params.evals.degree(); ++i)
    {
        for (uint64_t j = 0; j < nrowsPack; ++j)
        {
            evals_aux[(i * FIELD_EXTENSION) * nrowsPack + j] = params.evals[i][0];
            evals_aux[(i * FIELD_EXTENSION + 1) * nrowsPack + j] = params.evals[i][1];
            evals_aux[(i * FIELD_EXTENSION + 2) * nrowsPack + j] = params.evals[i][2];
        }
    }
    CHECKCUDAERR(hipMalloc((void **)&(stepPointers_h.evals_d), params.evals.degree() * FIELD_EXTENSION * nrowsPack * sizeof(gl64_t)));
    CHECKCUDAERR(hipMemcpy(stepPointers_h.evals_d, evals_aux, params.evals.degree() * FIELD_EXTENSION * nrowsPack * sizeof(gl64_t), hipMemcpyHostToDevice));
    delete[] evals_aux;

    CHECKCUDAERR(hipMalloc((void**)&(stepPointers_h.x_n_d), params.x_n.degree()*sizeof(gl64_t)));
    CHECKCUDAERR(hipMemcpy(stepPointers_h.x_n_d, params.x_n.address(), params.x_n.degree()*sizeof(gl64_t), hipMemcpyHostToDevice));

    CHECKCUDAERR(hipMalloc((void**)&(stepPointers_h.x_2ns_d), params.x_2ns.degree()*sizeof(gl64_t)));
    CHECKCUDAERR(hipMemcpy(stepPointers_h.x_2ns_d, params.x_2ns.address(), params.x_2ns.degree()*sizeof(gl64_t), hipMemcpyHostToDevice));

    CHECKCUDAERR(hipMalloc((void**)&(stepPointers_h.zi_d), params.zi.degree()*sizeof(gl64_t)));
    CHECKCUDAERR(hipMemcpy(stepPointers_h.zi_d, params.zi.address(), params.zi.degree()*sizeof(gl64_t), hipMemcpyHostToDevice));

    CHECKCUDAERR(hipMalloc((void**)&(stepPointers_h.xDivXSubXi_d), params.xDivXSubXi.degree()*params.xDivXSubXi.dim()*sizeof(gl64_t)));
    CHECKCUDAERR(hipMemcpy(stepPointers_h.xDivXSubXi_d, params.xDivXSubXi.address(), params.xDivXSubXi.degree()*params.xDivXSubXi.dim()*sizeof(gl64_t), hipMemcpyHostToDevice));

    /*
        temporal buffers
    */    

    stepPointers_h.dimBufferT = 2 * nCols * nrowsPack;
    CHECKCUDAERR(hipMalloc((void **)&(stepPointers_h.bufferT_d), stepPointers_h.dimBufferT * nstreams * sizeof(gl64_t)));
    
    stepPointers_h.dimBufferPols = 0;
    uint64_t nStages = 3;
    for (uint64_t s = 1; s <= nStages; ++s){
        stepPointers_h.dimBufferPols += nColsStages[s];
    }
    if(parserParams.stage==5){
        stepPointers_h.dimBufferPols += nColsStages[nStages + 1];
    }
    stepPointers_h.dimBufferPols += nColsStages[10]; //for the store
    stepPointers_h.dimBufferPols = stepPointers_h.dimBufferPols * (nrowsPack+nextStride);
    CHECKCUDAERR(hipMalloc((void **)&(stepPointers_h.bufferPols_d), stepPointers_h.dimBufferPols * nstreams * sizeof(gl64_t)));

    stepPointers_h.dimBufferConsts = starkInfo.nConstants * (nrowsPack+nextStride);
    CHECKCUDAERR(hipMalloc((void **)&(stepPointers_h.bufferConsts_d), stepPointers_h.dimBufferConsts * nstreams * sizeof(gl64_t)));
    
    stepPointers_h.dimTmp1 = parserParams.nTemp1 * nrowsPack;
    CHECKCUDAERR(hipMalloc((void **)&(stepPointers_h.tmp1_d), stepPointers_h.dimTmp1 * nstreams * sizeof(gl64_t)));
    
    stepPointers_h.dimTmp3 = parserParams.nTemp3 * nrowsPack * FIELD_EXTENSION;
    CHECKCUDAERR(hipMalloc((void **)&(stepPointers_h.tmp3_d), stepPointers_h.dimTmp3 * nstreams * sizeof(gl64_t)));

    /*
        copy to device
    */
    CHECKCUDAERR(hipMalloc((void **)&(stepPointers_d), sizeof(StepsPointers)));
    CHECKCUDAERR(hipMemcpy(stepPointers_d, &stepPointers_h, sizeof(StepsPointers), hipMemcpyHostToDevice));
}

__global__ void myadd(StepsPointers *in) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < 12) {
        in->nColsStages_d[i]++;
    }
}

void CHelpersStepsGPU::loadData(StarkInfo &starkInfo, StepsParams &params, ParserArgs &parserArgs, ParserParams &parserParams, uint64_t row, hipStream_t& stream){

    bool domainExtended = parserParams.stage > 3 ? true : false;
    uint32_t iStream = (row / nrowsPack) % nstreams; 
    gl64_t *bufferConsts_d = &stepPointers_h.bufferConsts_d[stepPointers_h.dimBufferConsts * iStream];
    gl64_t *bufferPols_d = &stepPointers_h.bufferPols_d[stepPointers_h.dimBufferPols * iStream];
    ConstantPolsStarks *constPols = domainExtended ? params.pConstPols2ns : params.pConstPols;


    CHECKCUDAERR(hipMemcpyAsync(bufferConsts_d, &(((Goldilocks::Element *)constPols->address())[row * starkInfo.nConstants]), stepPointers_h.dimBufferConsts*sizeof(Goldilocks::Element), hipMemcpyHostToDevice, stream));
    
    uint64_t nStages=3; // do I relly need to copy all
    uint64_t offset_pols_d=0;
    for (uint64_t s = 1; s <= nStages; ++s) 
    {
        
        uint64_t offset_pols_h = offsetsStages[s] + row * nColsStages[s];
        uint64_t size_copy = nColsStages[s]*(nrowsPack+stepPointers_h.nextStride);
        CHECKCUDAERR(hipMemcpyAsync(&(bufferPols_d[offset_pols_d]), &(params.pols[offset_pols_h]), size_copy*sizeof(Goldilocks::Element), hipMemcpyHostToDevice, stream));
        offset_pols_d += size_copy;
    }
    if (parserParams.stage == 5){

        uint64_t offset_pols_h = offsetsStages[nStages + 1] + row * nColsStages[nStages + 1];
        uint64_t size_copy = nColsStages[nStages + 1]*(nrowsPack+stepPointers_h.nextStride);
        CHECKCUDAERR(hipMemcpyAsync(&(bufferPols_d[offset_pols_d]), &(params.pols[offset_pols_h]), size_copy*sizeof(Goldilocks::Element), hipMemcpyHostToDevice, stream));
    }
}

void CHelpersStepsGPU::storeData(StarkInfo &starkInfo, StepsParams &params, ParserArgs &parserArgs, ParserParams &parserParams, uint64_t row, hipStream_t& stream){

    uint32_t iStream = (row / nrowsPack) % nstreams; 
    bool domainExtended = parserParams.stage > 3 ? true : false;
    gl64_t *bufferPols_d = &stepPointers_h.bufferPols_d[stepPointers_h.dimBufferPols * iStream];
    
    if (!domainExtended){
        uint64_t nStages=3; // do I relly need to copy all
        uint64_t offset_pols_d=0;
        for (uint64_t s = 2; s <= nStages + 1; ++s) //optimize copies that can be avoided...
        {
            
            uint64_t offset_pols_h = offsetsStages[s] + row * nColsStages[s];
            uint64_t size_copy = nColsStages[s]*nrowsPack;

            CHECKCUDAERR(hipMemcpyAsync(&(params.pols[offset_pols_h]), &(bufferPols_d[offset_pols_d]), size_copy*sizeof(Goldilocks::Element), hipMemcpyDeviceToHost, stream));
            offset_pols_d += size_copy;
        }
    }else{
        uint64_t size_copy = nColsStages[10]*nrowsPack;
        gl64_t *bufferPols_ = &(stepPointers_h.bufferPols_d[(iStream+1) * stepPointers_h.dimBufferPols-size_copy]); //data available at the end
        CHECKCUDAERR(hipMemcpyAsync(&(params.pols[offsetsStages[10] + row * nColsStages[10]]), bufferPols_, size_copy*sizeof(Goldilocks::Element), hipMemcpyDeviceToHost, stream));
    }
}

void CHelpersStepsGPU::calculateExpressions(StarkInfo &starkInfo, StepsParams &params, ParserArgs &parserArgs, ParserParams &parserParams)
{
    nrowsPack = 64;
    nstreams = 16;
    int nDevices;    
    CHECKCUDAERR(hipGetDeviceCount(&nDevices));


    bool domainExtended = parserParams.stage > 3 ? true : false;
    uint64_t domainSize = domainExtended ? 1 << starkInfo.starkStruct.nBitsExt : 1 << starkInfo.starkStruct.nBits;
    uint64_t nextStride = domainExtended ? 1 << (starkInfo.starkStruct.nBitsExt - starkInfo.starkStruct.nBits) : 1;

    // checka
     if(domainSize % nrowsPack != 0){
       zklog.error("nrowsPack should divide domainSize");
       exitProcess();
    }
    if(nrowsPack <= nextStride){
        zklog.error("nrowsPack should be greater than nextStride");
        exitProcess();
    }

    // The last pack of rows are solved uwing the chelpers_pack
    
    CHelpersStepsPack chelpersPack;
    chelpersPack.calculateExpressionsRows(starkInfo, params, parserArgs, parserParams, domainSize-nrowsPack, domainSize);

    //Rest of packs are copmuted in the GPU...
    dataSetup(starkInfo, params, parserArgs, parserParams);

    uint64_t mysize = nColsStages.size();
    printf("nColsStages size%lu:\n", mysize);
    for (uint64_t i=0; i<mysize; i++) {
        printf("%lu\n", nColsStages[i]);
    }
    uint64_t *mybuffer = (uint64_t *)malloc(mysize * sizeof(uint64_t));
    CHECKCUDAERR(hipMemcpy(mybuffer, stepPointers_h.nColsStages_d, mysize * sizeof(uint64_t), hipMemcpyDeviceToHost));
    printf("stepPointers_h.nColsStages_d:\n");
    for (uint64_t i=0; i<mysize; i++) {
        printf("%lu\n", mybuffer[i]);
    }

    myadd<<<1, 64>>>(stepPointers_d);

    StepsPointers *tmpPointer = (StepsPointers *)malloc(sizeof(StepsPointers));
    CHECKCUDAERR(hipMemcpy(tmpPointer, stepPointers_d, sizeof(StepsPointers), hipMemcpyDeviceToHost));

    CHECKCUDAERR(hipMemcpy(mybuffer, tmpPointer->nColsStages_d, mysize * sizeof(uint64_t), hipMemcpyDeviceToHost));
    printf("stepPointers_h.nColsStages_d:\n");
    for (uint64_t i=0; i<mysize; i++) {
        printf("%lu\n", mybuffer[i]);
    }

    printf("nCols:%lu\n", nCols);
    printf("nrowsPack:%lu\n", nrowsPack);
    printf("domainSize:%lu\n", stepPointers_h.domainSize);
    printf("nConstants:%lu\n", stepPointers_h.nConstants);
    printf("nextStride:%lu\n", stepPointers_h.nextStride);

    printf("dimBufferT:%u\n", stepPointers_h.dimBufferT);
    printf("dimBufferPols:%u\n", stepPointers_h.dimBufferPols);
    printf("dimBufferConsts:%u\n", stepPointers_h.dimBufferConsts);
    printf("dimTmp1:%u\n", stepPointers_h.dimTmp1);
    printf("dimTmp3:%u\n", stepPointers_h.dimTmp3);

    printf("nConstants:%lu\n", starkInfo.nConstants);
    printf("nPublics:%lu\n", starkInfo.nPublics);
    printf("nCm1:%lu\n", starkInfo.nCm1);
    printf("nCm2:%lu\n", starkInfo.nCm2);
    printf("nCm3:%lu\n", starkInfo.nCm3);
    printf("nCm4:%lu\n", starkInfo.nCm4);
    printf("qDeg:%lu\n", starkInfo.qDeg);
    printf("qDim:%lu\n", starkInfo.qDim);
    printf("friExpId:%lu\n", starkInfo.friExpId);
    printf("merkleTreeArity:%lu\n", starkInfo.merkleTreeArity);

    hipStream_t *streams = new hipStream_t[nstreams];
    for (int i = 0; i < nstreams; i++)
    {
        hipStreamCreate(&streams[i]);
    }

    for (uint64_t i = 0; i < domainSize-nrowsPack; i += nrowsPack)
    {
        uint32_t iStream = (i / nrowsPack) % nstreams; 
        loadData(starkInfo, params, parserArgs, parserParams, i, streams[iStream]);
        _transposeToBuffer<<<1, nrowsPack, 0, streams[iStream]>>>(stepPointers_d, i, parserParams.stage, domainExtended, iStream);
        _packComputation<<<1, nrowsPack, 0, streams[iStream]>>>(stepPointers_d, domainSize, parserParams.nOps, parserParams.nArgs, iStream);
        _transposeFromBuffer<<<1, nrowsPack, 0, streams[iStream]>>>(stepPointers_d, i, parserParams.stage, domainExtended, iStream);
        storeData(starkInfo, params, parserArgs, parserParams, i, streams[iStream]);    
    }

    //
    // Synchronize and Destroy Streams and free memory
    //
    for (int i = 0; i < nstreams; i++)
    {
        hipStreamSynchronize(streams[i]);
        hipStreamDestroy(streams[i]);
    }
    freePointers();
   
    delete[] streams;
}

void CHelpersStepsGPU::freePointers()
{
    hipFree(stepPointers_h.nColsStages_d);
    hipFree(stepPointers_h.nColsStagesAcc_d);
    hipFree(stepPointers_h.ops_d);
    hipFree(stepPointers_h.args_d);
    hipFree(stepPointers_h.numbers_d);
    hipFree(stepPointers_h.challenges_d);
    hipFree(stepPointers_h.challenges_ops_d);
    hipFree(stepPointers_h.publics_d);
    hipFree(stepPointers_h.evals_d);
    hipFree(stepPointers_h.x_n_d);
    hipFree(stepPointers_h.x_2ns_d);
    hipFree(stepPointers_h.zi_d);
    hipFree(stepPointers_h.xDivXSubXi_d);
    hipFree(stepPointers_h.bufferT_d);
    hipHostFree(stepPointers_h.bufferPols_d);
    hipHostFree(stepPointers_h.bufferConsts_d);
    hipFree(stepPointers_h.tmp1_d);
    hipFree(stepPointers_h.tmp3_d);
    hipFree(stepPointers_d);
}

__global__ void _packComputation(StepsPointers *stepPointers_d, uint32_t N, uint32_t nOps, uint32_t nArgs, uint32_t stream)
{

    uint64_t i_args = 0;
    gl64_t *bufferT_ = &(stepPointers_d->bufferT_d[stream * stepPointers_d->dimBufferT]);
    gl64_t *tmp1 = &(stepPointers_d->tmp1_d[stream * stepPointers_d->dimTmp1]);
    gl64_t *tmp3 = &(stepPointers_d->tmp3_d[stream * stepPointers_d->dimTmp3]);
    uint32_t *ops_ = stepPointers_d->ops_d;
    uint32_t *args_ = stepPointers_d->args_d;
    
    for (uint64_t kk = 0; kk < nOps; ++kk)
    {
        switch (ops_[kk])
        {
        case 0:
        {
            // COPY commit1 to commit1
            gl64_t::copy_gpu(&bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args]] + args_[i_args + 1]) * blockDim.x], &bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 2]] + args_[i_args + 3]) * blockDim.x]);
            i_args += 4;
            break;
        }
        case 1:
        {
            // OPERATION WITH DEST: commit1 - SRC0: commit1 - SRC1: commit1
            gl64_t::op_gpu(args_[i_args], &bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 1]] + args_[i_args + 2]) * blockDim.x], &bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 3]] + args_[i_args + 4]) * blockDim.x], &bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 5]] + args_[i_args + 6]) * blockDim.x]);
            i_args += 7;
            break;
        }
        case 2:
        {
            // OPERATION WITH DEST: commit1 - SRC0: commit1 - SRC1: tmp1
            gl64_t::op_gpu(args_[i_args], &bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 1]] + args_[i_args + 2]) * blockDim.x], &bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 3]] + args_[i_args + 4]) * blockDim.x], &tmp1[args_[i_args + 5] * blockDim.x]);
            i_args += 6;
            break;
        }
        case 3:
        {
            // OPERATION WITH DEST: commit1 - SRC0: commit1 - SRC1: public
            gl64_t::op_gpu(args_[i_args], &bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 1]] + args_[i_args + 2]) * blockDim.x], &bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 3]] + args_[i_args + 4]) * blockDim.x], &stepPointers_d->publics_d[args_[i_args + 5] * blockDim.x]);
            i_args += 6;
            break;
        }
        case 4:
        {
            // OPERATION WITH DEST: commit1 - SRC0: commit1 - SRC1: number
            gl64_t::op_gpu(args_[i_args], &bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 1]] + args_[i_args + 2]) * blockDim.x], &bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 3]] + args_[i_args + 4]) * blockDim.x], &stepPointers_d->numbers_d[args_[i_args + 5] * blockDim.x]);
            i_args += 6;
            break;
        }
        case 5:
        {
            // COPY tmp1 to commit1
            gl64_t::copy_gpu(&bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args]] + args_[i_args + 1]) * blockDim.x], &tmp1[args_[i_args + 2] * blockDim.x]);
            i_args += 3;
            break;
        }
        case 6:
        {
            // OPERATION WITH DEST: commit1 - SRC0: tmp1 - SRC1: tmp1
            gl64_t::op_gpu(args_[i_args], &bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 1]] + args_[i_args + 2]) * blockDim.x], &tmp1[args_[i_args + 3] * blockDim.x], &tmp1[args_[i_args + 4] * blockDim.x]);
            i_args += 5;
            break;
        }
        case 7:
        {
            // OPERATION WITH DEST: commit1 - SRC0: tmp1 - SRC1: public
            gl64_t::op_gpu(args_[i_args], &bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 1]] + args_[i_args + 2]) * blockDim.x], &tmp1[args_[i_args + 3] * blockDim.x], &stepPointers_d->publics_d[args_[i_args + 4] * blockDim.x]);
            i_args += 5;
            break;
        }
        case 8:
        {
            // OPERATION WITH DEST: commit1 - SRC0: tmp1 - SRC1: number
            gl64_t::op_gpu(args_[i_args], &bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 1]] + args_[i_args + 2]) * blockDim.x], &tmp1[args_[i_args + 3] * blockDim.x], &stepPointers_d->numbers_d[args_[i_args + 4] * blockDim.x]);
            i_args += 5;
            break;
        }
        case 9:
        {
            // COPY public to commit1
            gl64_t::copy_gpu(&bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args]] + args_[i_args + 1]) * blockDim.x], &stepPointers_d->publics_d[args_[i_args + 2] * blockDim.x]);
            i_args += 3;
            break;
        }
        case 10:
        {
            // OPERATION WITH DEST: commit1 - SRC0: public - SRC1: public
            gl64_t::op_gpu(args_[i_args], &bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 1]] + args_[i_args + 2]) * blockDim.x], &stepPointers_d->publics_d[args_[i_args + 3] * blockDim.x], &stepPointers_d->publics_d[args_[i_args + 4] * blockDim.x]);
            i_args += 5;
            break;
        }
        case 11:
        {
            // OPERATION WITH DEST: commit1 - SRC0: public - SRC1: number
            gl64_t::op_gpu(args_[i_args], &bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 1]] + args_[i_args + 2]) * blockDim.x], &stepPointers_d->publics_d[args_[i_args + 3] * blockDim.x], &stepPointers_d->numbers_d[args_[i_args + 4] * blockDim.x]);
            i_args += 5;
            break;
        }
        case 12:
        {
            // COPY number to commit1
            gl64_t::copy_gpu(&bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args]] + args_[i_args + 1]) * blockDim.x], &stepPointers_d->numbers_d[args_[i_args + 2] * blockDim.x]);
            i_args += 3;
            break;
        }
        case 13:
        {
            // OPERATION WITH DEST: commit1 - SRC0: number - SRC1: number
            gl64_t::op_gpu(args_[i_args], &bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 1]] + args_[i_args + 2]) * blockDim.x], &stepPointers_d->numbers_d[args_[i_args + 3] * blockDim.x], &stepPointers_d->numbers_d[args_[i_args + 4] * blockDim.x]);
            i_args += 5;
            break;
        }
        case 14:
        {
            // COPY commit1 to tmp1
            gl64_t::copy_gpu(&tmp1[args_[i_args] * blockDim.x], &bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 1]] + args_[i_args + 2]) * blockDim.x]);
            i_args += 3;
            break;
        }
        case 15:
        {
            // OPERATION WITH DEST: tmp1 - SRC0: commit1 - SRC1: commit1
            gl64_t::op_gpu(args_[i_args], &tmp1[args_[i_args + 1] * blockDim.x], &bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 2]] + args_[i_args + 3]) * blockDim.x], &bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 4]] + args_[i_args + 5]) * blockDim.x]);
            i_args += 6;
            break;
        }
        case 16:
        {
            // OPERATION WITH DEST: tmp1 - SRC0: commit1 - SRC1: tmp1
            gl64_t::op_gpu(args_[i_args], &tmp1[args_[i_args + 1] * blockDim.x], &bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 2]] + args_[i_args + 3]) * blockDim.x], &tmp1[args_[i_args + 4] * blockDim.x]);
            i_args += 5;
            break;
        }
        case 17:
        {
            // OPERATION WITH DEST: tmp1 - SRC0: commit1 - SRC1: public
            gl64_t::op_gpu(args_[i_args], &tmp1[args_[i_args + 1] * blockDim.x], &bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 2]] + args_[i_args + 3]) * blockDim.x], &stepPointers_d->publics_d[args_[i_args + 4] * blockDim.x]);
            i_args += 5;
            break;
        }
        case 18:
        {
            // OPERATION WITH DEST: tmp1 - SRC0: commit1 - SRC1: number
            gl64_t::op_gpu(args_[i_args], &tmp1[args_[i_args + 1] * blockDim.x], &bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 2]] + args_[i_args + 3]) * blockDim.x], &stepPointers_d->numbers_d[args_[i_args + 4] * blockDim.x]);
            i_args += 5;
            break;
        }
        case 19:
        {
            // COPY tmp1 to tmp1
            gl64_t::copy_gpu(&tmp1[args_[i_args] * blockDim.x], &tmp1[args_[i_args + 1] * blockDim.x]);
            i_args += 2;
            break;
        }
        case 20:
        {
            // OPERATION WITH DEST: tmp1 - SRC0: tmp1 - SRC1: tmp1
            gl64_t::op_gpu(args_[i_args], &tmp1[args_[i_args + 1] * blockDim.x], &tmp1[args_[i_args + 2] * blockDim.x], &tmp1[args_[i_args + 3] * blockDim.x]);
            i_args += 4;
            break;
        }
        case 21:
        {
            // OPERATION WITH DEST: tmp1 - SRC0: tmp1 - SRC1: public
            gl64_t::op_gpu(args_[i_args], &tmp1[args_[i_args + 1] * blockDim.x], &tmp1[args_[i_args + 2] * blockDim.x], &stepPointers_d->publics_d[args_[i_args + 3] * blockDim.x]);
            i_args += 4;
            break;
        }
        case 22:
        {
            // OPERATION WITH DEST: tmp1 - SRC0: tmp1 - SRC1: number
            gl64_t::op_gpu(args_[i_args], &tmp1[args_[i_args + 1] * blockDim.x], &tmp1[args_[i_args + 2] * blockDim.x], &stepPointers_d->numbers_d[args_[i_args + 3] * blockDim.x]);
            i_args += 4;
            break;
        }
        case 23:
        {
            // COPY public to tmp1
            gl64_t::copy_gpu(&tmp1[args_[i_args] * blockDim.x], &stepPointers_d->publics_d[args_[i_args + 1] * blockDim.x]);
            i_args += 2;
            break;
        }
        case 24:
        {
            // OPERATION WITH DEST: tmp1 - SRC0: public - SRC1: public
            gl64_t::op_gpu(args_[i_args], &tmp1[args_[i_args + 1] * blockDim.x], &stepPointers_d->publics_d[args_[i_args + 2] * blockDim.x], &stepPointers_d->publics_d[args_[i_args + 3] * blockDim.x]);
            i_args += 4;
            break;
        }
        case 25:
        {
            // OPERATION WITH DEST: tmp1 - SRC0: public - SRC1: number
            gl64_t::op_gpu(args_[i_args], &tmp1[args_[i_args + 1] * blockDim.x], &stepPointers_d->publics_d[args_[i_args + 2] * blockDim.x], &stepPointers_d->numbers_d[args_[i_args + 3] * blockDim.x]);
            i_args += 4;
            break;
        }
        case 26:
        {
            // COPY number to tmp1
            gl64_t::copy_gpu(&tmp1[args_[i_args] * blockDim.x], &stepPointers_d->numbers_d[args_[i_args + 1] * blockDim.x]);
            i_args += 2;
            break;
        }
        case 27:
        {
            // OPERATION WITH DEST: tmp1 - SRC0: number - SRC1: number
            gl64_t::op_gpu(args_[i_args], &tmp1[args_[i_args + 1] * blockDim.x], &stepPointers_d->numbers_d[args_[i_args + 2] * blockDim.x], &stepPointers_d->numbers_d[args_[i_args + 3] * blockDim.x]);
            i_args += 4;
            break;
        }
        case 28:
        {
            // OPERATION WITH DEST: commit3 - SRC0: commit3 - SRC1: commit1
            Goldilocks3GPU::op_31_gpu(args_[i_args], &bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 1]] + args_[i_args + 2]) * blockDim.x], &bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 3]] + args_[i_args + 4]) * blockDim.x], &bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 5]] + args_[i_args + 6]) * blockDim.x]);
            i_args += 7;
            break;
        }
        case 29:
        {
            // OPERATION WITH DEST: commit3 - SRC0: commit3 - SRC1: tmp1
            Goldilocks3GPU::op_31_gpu(args_[i_args], &bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 1]] + args_[i_args + 2]) * blockDim.x], &bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 3]] + args_[i_args + 4]) * blockDim.x], &tmp1[args_[i_args + 5] * blockDim.x]);
            i_args += 6;
            break;
        }
        case 30:
        {
            // OPERATION WITH DEST: commit3 - SRC0: commit3 - SRC1: public
            Goldilocks3GPU::op_31_gpu(args_[i_args], &bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 1]] + args_[i_args + 2]) * blockDim.x], &bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 3]] + args_[i_args + 4]) * blockDim.x], &stepPointers_d->publics_d[args_[i_args + 5] * blockDim.x]);
            i_args += 6;
            break;
        }
        case 31:
        {
            // OPERATION WITH DEST: commit3 - SRC0: commit3 - SRC1: number
            Goldilocks3GPU::op_31_gpu(args_[i_args], &bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 1]] + args_[i_args + 2]) * blockDim.x], &bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 3]] + args_[i_args + 4]) * blockDim.x], &stepPointers_d->numbers_d[args_[i_args + 5] * blockDim.x]);
            i_args += 6;
            break;
        }
        case 32:
        {
            // OPERATION WITH DEST: commit3 - SRC0: tmp3 - SRC1: commit1
            Goldilocks3GPU::op_31_gpu(args_[i_args], &bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 1]] + args_[i_args + 2]) * blockDim.x], &tmp3[args_[i_args + 3] * blockDim.x * FIELD_EXTENSION], &bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 4]] + args_[i_args + 5]) * blockDim.x]);
            i_args += 6;
            break;
        }
        case 33:
        {
            // OPERATION WITH DEST: commit3 - SRC0: tmp3 - SRC1: tmp1
            Goldilocks3GPU::op_31_gpu(args_[i_args], &bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 1]] + args_[i_args + 2]) * blockDim.x], &tmp3[args_[i_args + 3] * blockDim.x * FIELD_EXTENSION], &tmp1[args_[i_args + 4] * blockDim.x]);
            i_args += 5;
            break;
        }
        case 34:
        {
            // OPERATION WITH DEST: commit3 - SRC0: tmp3 - SRC1: public
            Goldilocks3GPU::op_31_gpu(args_[i_args], &bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 1]] + args_[i_args + 2]) * blockDim.x], &tmp3[args_[i_args + 3] * blockDim.x * FIELD_EXTENSION], &stepPointers_d->publics_d[args_[i_args + 4] * blockDim.x]);
            i_args += 5;
            break;
        }
        case 35:
        {
            // OPERATION WITH DEST: commit3 - SRC0: tmp3 - SRC1: number
            Goldilocks3GPU::op_31_gpu(args_[i_args], &bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 1]] + args_[i_args + 2]) * blockDim.x], &tmp3[args_[i_args + 3] * blockDim.x * FIELD_EXTENSION], &stepPointers_d->numbers_d[args_[i_args + 4] * blockDim.x]);
            i_args += 5;
            break;
        }
        case 36:
        {
            // OPERATION WITH DEST: commit3 - SRC0: challenge - SRC1: commit1
            Goldilocks3GPU::op_31_gpu(args_[i_args], &bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 1]] + args_[i_args + 2]) * blockDim.x], &stepPointers_d->challenges_d[args_[i_args + 3] * FIELD_EXTENSION * blockDim.x], &bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 4]] + args_[i_args + 5]) * blockDim.x]);
            i_args += 6;
            break;
        }
        case 37:
        {
            // OPERATION WITH DEST: commit3 - SRC0: challenge - SRC1: tmp1
            Goldilocks3GPU::op_31_gpu(args_[i_args], &bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 1]] + args_[i_args + 2]) * blockDim.x], &stepPointers_d->challenges_d[args_[i_args + 3] * FIELD_EXTENSION * blockDim.x], &tmp1[args_[i_args + 4] * blockDim.x]);
            i_args += 5;
            break;
        }
        case 38:
        {
            // OPERATION WITH DEST: commit3 - SRC0: challenge - SRC1: public
            Goldilocks3GPU::op_31_gpu(args_[i_args], &bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 1]] + args_[i_args + 2]) * blockDim.x], &stepPointers_d->challenges_d[args_[i_args + 3] * FIELD_EXTENSION * blockDim.x], &stepPointers_d->publics_d[args_[i_args + 4] * blockDim.x]);
            i_args += 5;
            break;
        }
        case 39:
        {
            // OPERATION WITH DEST: commit3 - SRC0: challenge - SRC1: number
            Goldilocks3GPU::op_31_gpu(args_[i_args], &bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 1]] + args_[i_args + 2]) * blockDim.x], &stepPointers_d->challenges_d[args_[i_args + 3] * FIELD_EXTENSION * blockDim.x], &stepPointers_d->numbers_d[args_[i_args + 4] * blockDim.x]);
            i_args += 5;
            break;
        }
        case 40:
        {
            // COPY commit3 to commit3
            Goldilocks3GPU::copy_gpu(&bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args]] + args_[i_args + 1]) * blockDim.x], &bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 2]] + args_[i_args + 3]) * blockDim.x]);
            i_args += 4;
            break;
        }
        case 41:
        {
            // OPERATION WITH DEST: commit3 - SRC0: commit3 - SRC1: commit3
            Goldilocks3GPU::op_gpu(args_[i_args], &bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 1]] + args_[i_args + 2]) * blockDim.x], &bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 3]] + args_[i_args + 4]) * blockDim.x], &bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 5]] + args_[i_args + 6]) * blockDim.x]);
            i_args += 7;
            break;
        }
        case 42:
        {
            // OPERATION WITH DEST: commit3 - SRC0: commit3 - SRC1: tmp3
            Goldilocks3GPU::op_gpu(args_[i_args], &bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 1]] + args_[i_args + 2]) * blockDim.x], &bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 3]] + args_[i_args + 4]) * blockDim.x], &tmp3[args_[i_args + 5] * blockDim.x * FIELD_EXTENSION]);
            i_args += 6;
            break;
        }
        case 43:
        {
            // MULTIPLICATION WITH DEST: commit3 - SRC0: commit3 - SRC1: challenge
            Goldilocks3GPU::mul_gpu(&bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 1]] + args_[i_args + 2]) * blockDim.x], &bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 3]] + args_[i_args + 4]) * blockDim.x], &stepPointers_d->challenges_d[args_[i_args + 5] * FIELD_EXTENSION * blockDim.x], &stepPointers_d->challenges_ops_d[args_[i_args + 5] * FIELD_EXTENSION * blockDim.x]);
            i_args += 6;
            break;
        }
        case 44:
        {
            // OPERATION WITH DEST: commit3 - SRC0: commit3 - SRC1: challenge
            Goldilocks3GPU::op_gpu(args_[i_args], &bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 1]] + args_[i_args + 2]) * blockDim.x], &bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 3]] + args_[i_args + 4]) * blockDim.x], &stepPointers_d->challenges_d[args_[i_args + 5] * FIELD_EXTENSION * blockDim.x]);
            i_args += 6;
            break;
        }
        case 45:
        {
            // COPY tmp3 to commit3
            Goldilocks3GPU::copy_gpu(&bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args]] + args_[i_args + 1]) * blockDim.x], &tmp3[args_[i_args + 2] * blockDim.x * FIELD_EXTENSION]);
            i_args += 3;
            break;
        }
        case 46:
        {
            // OPERATION WITH DEST: commit3 - SRC0: tmp3 - SRC1: tmp3
            Goldilocks3GPU::op_gpu(args_[i_args], &bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 1]] + args_[i_args + 2]) * blockDim.x], &tmp3[args_[i_args + 3] * blockDim.x * FIELD_EXTENSION], &tmp3[args_[i_args + 4] * blockDim.x * FIELD_EXTENSION]);
            i_args += 5;
            break;
        }
        case 47:
        {
            // MULTIPLICATION WITH DEST: commit3 - SRC0: tmp3 - SRC1: challenge
            Goldilocks3GPU::mul_gpu(&bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 1]] + args_[i_args + 2]) * blockDim.x], &tmp3[args_[i_args + 3] * blockDim.x * FIELD_EXTENSION], &stepPointers_d->challenges_d[args_[i_args + 4] * FIELD_EXTENSION * blockDim.x], &stepPointers_d->challenges_ops_d[args_[i_args + 4] * FIELD_EXTENSION * blockDim.x]);
            i_args += 5;
            break;
        }
        case 48:
        {
            // OPERATION WITH DEST: commit3 - SRC0: tmp3 - SRC1: challenge
            Goldilocks3GPU::op_gpu(args_[i_args], &bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 1]] + args_[i_args + 2]) * blockDim.x], &tmp3[args_[i_args + 3] * blockDim.x * FIELD_EXTENSION], &stepPointers_d->challenges_d[args_[i_args + 4] * FIELD_EXTENSION * blockDim.x]);
            i_args += 5;
            break;
        }
        case 49:
        {
            // MULTIPLICATION WITH DEST: commit3 - SRC0: challenge - SRC1: challenge
            Goldilocks3GPU::mul_gpu(&bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 1]] + args_[i_args + 2]) * blockDim.x], &stepPointers_d->challenges_d[args_[i_args + 3] * FIELD_EXTENSION * blockDim.x], &stepPointers_d->challenges_d[args_[i_args + 4] * FIELD_EXTENSION * blockDim.x], &stepPointers_d->challenges_ops_d[args_[i_args + 4] * FIELD_EXTENSION * blockDim.x]);
            i_args += 5;
            break;
        }
        case 50:
        {
            // OPERATION WITH DEST: commit3 - SRC0: challenge - SRC1: challenge
            Goldilocks3GPU::op_gpu(args_[i_args], &bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 1]] + args_[i_args + 2]) * blockDim.x], &stepPointers_d->challenges_d[args_[i_args + 3] * FIELD_EXTENSION * blockDim.x], &stepPointers_d->challenges_d[args_[i_args + 4] * FIELD_EXTENSION * blockDim.x]);
            i_args += 5;
            break;
        }
        case 51:
        {
            // OPERATION WITH DEST: tmp3 - SRC0: commit3 - SRC1: commit1
            Goldilocks3GPU::op_31_gpu(args_[i_args], &tmp3[args_[i_args + 1] * blockDim.x * FIELD_EXTENSION], &bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 2]] + args_[i_args + 3]) * blockDim.x], &bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 4]] + args_[i_args + 5]) * blockDim.x]);
            i_args += 6;
            break;
        }
        case 52:
        {
            // OPERATION WITH DEST: tmp3 - SRC0: commit3 - SRC1: tmp1
            Goldilocks3GPU::op_31_gpu(args_[i_args], &tmp3[args_[i_args + 1] * blockDim.x * FIELD_EXTENSION], &bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 2]] + args_[i_args + 3]) * blockDim.x], &tmp1[args_[i_args + 4] * blockDim.x]);
            i_args += 5;
            break;
        }
        case 53:
        {
            // OPERATION WITH DEST: tmp3 - SRC0: commit3 - SRC1: public
            Goldilocks3GPU::op_31_gpu(args_[i_args], &tmp3[args_[i_args + 1] * blockDim.x * FIELD_EXTENSION], &bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 2]] + args_[i_args + 3]) * blockDim.x], &stepPointers_d->publics_d[args_[i_args + 4] * blockDim.x]);
            i_args += 5;
            break;
        }
        case 54:
        {
            // OPERATION WITH DEST: tmp3 - SRC0: commit3 - SRC1: number
            Goldilocks3GPU::op_31_gpu(args_[i_args], &tmp3[args_[i_args + 1] * blockDim.x * FIELD_EXTENSION], &bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 2]] + args_[i_args + 3]) * blockDim.x], &stepPointers_d->numbers_d[args_[i_args + 4] * blockDim.x]);
            i_args += 5;
            break;
        }
        case 55:
        {
            // OPERATION WITH DEST: tmp3 - SRC0: tmp3 - SRC1: commit1
            Goldilocks3GPU::op_31_gpu(args_[i_args], &tmp3[args_[i_args + 1] * blockDim.x * FIELD_EXTENSION], &tmp3[args_[i_args + 2] * blockDim.x * FIELD_EXTENSION], &bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 3]] + args_[i_args + 4]) * blockDim.x]);
            i_args += 5;
            break;
        }
        case 56:
        {
            // OPERATION WITH DEST: tmp3 - SRC0: tmp3 - SRC1: tmp1
            Goldilocks3GPU::op_31_gpu(args_[i_args], &tmp3[args_[i_args + 1] * blockDim.x * FIELD_EXTENSION], &tmp3[args_[i_args + 2] * blockDim.x * FIELD_EXTENSION], &tmp1[args_[i_args + 3] * blockDim.x]);
            i_args += 4;
            break;
        }
        case 57:
        {
            // OPERATION WITH DEST: tmp3 - SRC0: tmp3 - SRC1: public
            Goldilocks3GPU::op_31_gpu(args_[i_args], &tmp3[args_[i_args + 1] * blockDim.x * FIELD_EXTENSION], &tmp3[args_[i_args + 2] * blockDim.x * FIELD_EXTENSION], &stepPointers_d->publics_d[args_[i_args + 3] * blockDim.x]);
            i_args += 4;
            break;
        }
        case 58:
        {
            // OPERATION WITH DEST: tmp3 - SRC0: tmp3 - SRC1: number
            Goldilocks3GPU::op_31_gpu(args_[i_args], &tmp3[args_[i_args + 1] * blockDim.x * FIELD_EXTENSION], &tmp3[args_[i_args + 2] * blockDim.x * FIELD_EXTENSION], &stepPointers_d->numbers_d[args_[i_args + 3] * blockDim.x]);
            i_args += 4;
            break;
        }
        case 59:
        {
            // OPERATION WITH DEST: tmp3 - SRC0: challenge - SRC1: commit1
            Goldilocks3GPU::op_31_gpu(args_[i_args], &tmp3[args_[i_args + 1] * blockDim.x * FIELD_EXTENSION], &stepPointers_d->challenges_d[args_[i_args + 2] * FIELD_EXTENSION * blockDim.x], &bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 3]] + args_[i_args + 4]) * blockDim.x]);
            i_args += 5;
            break;
        }
        case 60:
        {
            // OPERATION WITH DEST: tmp3 - SRC0: challenge - SRC1: tmp1
            Goldilocks3GPU::op_31_gpu(args_[i_args], &tmp3[args_[i_args + 1] * blockDim.x * FIELD_EXTENSION], &stepPointers_d->challenges_d[args_[i_args + 2] * FIELD_EXTENSION * blockDim.x], &tmp1[args_[i_args + 3] * blockDim.x]);
            i_args += 4;
            break;
        }
        case 61:
        {
            // OPERATION WITH DEST: tmp3 - SRC0: challenge - SRC1: public
            Goldilocks3GPU::op_31_gpu(args_[i_args], &tmp3[args_[i_args + 1] * blockDim.x * FIELD_EXTENSION], &stepPointers_d->challenges_d[args_[i_args + 2] * FIELD_EXTENSION * blockDim.x], &stepPointers_d->publics_d[args_[i_args + 3] * blockDim.x]);
            i_args += 4;
            break;
        }
        case 62:
        {
            // OPERATION WITH DEST: tmp3 - SRC0: challenge - SRC1: number
            Goldilocks3GPU::op_31_gpu(args_[i_args], &tmp3[args_[i_args + 1] * blockDim.x * FIELD_EXTENSION], &stepPointers_d->challenges_d[args_[i_args + 2] * FIELD_EXTENSION * blockDim.x], &stepPointers_d->numbers_d[args_[i_args + 3] * blockDim.x]);
            i_args += 4;
            break;
        }
        case 63:
        {
            // COPY commit3 to tmp3
            Goldilocks3GPU::copy_gpu(&tmp3[args_[i_args] * blockDim.x * FIELD_EXTENSION], &bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 1]] + args_[i_args + 2]) * blockDim.x]);
            i_args += 3;
            break;
        }
        case 64:
        {
            // OPERATION WITH DEST: tmp3 - SRC0: commit3 - SRC1: commit3
            Goldilocks3GPU::op_gpu(args_[i_args], &tmp3[args_[i_args + 1] * blockDim.x * FIELD_EXTENSION], &bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 2]] + args_[i_args + 3]) * blockDim.x], &bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 4]] + args_[i_args + 5]) * blockDim.x]);
            i_args += 6;
            break;
        }
        case 65:
        {
            // OPERATION WITH DEST: tmp3 - SRC0: commit3 - SRC1: tmp3
            Goldilocks3GPU::op_gpu(args_[i_args], &tmp3[args_[i_args + 1] * blockDim.x * FIELD_EXTENSION], &bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 2]] + args_[i_args + 3]) * blockDim.x], &tmp3[args_[i_args + 4] * blockDim.x * FIELD_EXTENSION]);
            i_args += 5;
            break;
        }
        case 66:
        {
            // MULTIPLICATION WITH DEST: tmp3 - SRC0: commit3 - SRC1: challenge
            Goldilocks3GPU::mul_gpu(&tmp3[args_[i_args + 1] * blockDim.x * FIELD_EXTENSION], &bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 2]] + args_[i_args + 3]) * blockDim.x], &stepPointers_d->challenges_d[args_[i_args + 4] * FIELD_EXTENSION * blockDim.x], &stepPointers_d->challenges_ops_d[args_[i_args + 4] * FIELD_EXTENSION * blockDim.x]);
            i_args += 5;
            break;
        }
        case 67:
        {
            // OPERATION WITH DEST: tmp3 - SRC0: commit3 - SRC1: challenge
            Goldilocks3GPU::op_gpu(args_[i_args], &tmp3[args_[i_args + 1] * blockDim.x * FIELD_EXTENSION], &bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 2]] + args_[i_args + 3]) * blockDim.x], &stepPointers_d->challenges_d[args_[i_args + 4] * FIELD_EXTENSION * blockDim.x]);
            i_args += 5;
            break;
        }
        case 68:
        {
            // COPY tmp3 to tmp3
            Goldilocks3GPU::copy_gpu(&tmp3[args_[i_args] * blockDim.x * FIELD_EXTENSION], &tmp3[args_[i_args + 1] * blockDim.x * FIELD_EXTENSION]);
            i_args += 2;
            break;
        }
        case 69:
        {
            // OPERATION WITH DEST: tmp3 - SRC0: tmp3 - SRC1: tmp3
            Goldilocks3GPU::op_gpu(args_[i_args], &tmp3[args_[i_args + 1] * blockDim.x * FIELD_EXTENSION], &tmp3[args_[i_args + 2] * blockDim.x * FIELD_EXTENSION], &tmp3[args_[i_args + 3] * blockDim.x * FIELD_EXTENSION]);
            i_args += 4;
            break;
        }
        case 70:
        {
            // MULTIPLICATION WITH DEST: tmp3 - SRC0: tmp3 - SRC1: challenge
            Goldilocks3GPU::mul_gpu(&tmp3[args_[i_args + 1] * blockDim.x * FIELD_EXTENSION], &tmp3[args_[i_args + 2] * blockDim.x * FIELD_EXTENSION], &stepPointers_d->challenges_d[args_[i_args + 3] * FIELD_EXTENSION * blockDim.x], &stepPointers_d->challenges_ops_d[args_[i_args + 3] * FIELD_EXTENSION * blockDim.x]);
            i_args += 4;
            break;
        }
        case 71:
        {
            // OPERATION WITH DEST: tmp3 - SRC0: tmp3 - SRC1: challenge
            Goldilocks3GPU::op_gpu(args_[i_args], &tmp3[args_[i_args + 1] * blockDim.x * FIELD_EXTENSION], &tmp3[args_[i_args + 2] * blockDim.x * FIELD_EXTENSION], &stepPointers_d->challenges_d[args_[i_args + 3] * FIELD_EXTENSION * blockDim.x]);
            i_args += 4;
            break;
        }
        case 72:
        {
            // MULTIPLICATION WITH DEST: tmp3 - SRC0: challenge - SRC1: challenge
            Goldilocks3GPU::mul_gpu(&tmp3[args_[i_args + 1] * blockDim.x * FIELD_EXTENSION], &stepPointers_d->challenges_d[args_[i_args + 2] * FIELD_EXTENSION * blockDim.x], &stepPointers_d->challenges_d[args_[i_args + 3] * FIELD_EXTENSION * blockDim.x], &stepPointers_d->challenges_ops_d[args_[i_args + 3] * FIELD_EXTENSION * blockDim.x]);
            i_args += 4;
            break;
        }
        case 73:
        {
            // OPERATION WITH DEST: tmp3 - SRC0: challenge - SRC1: challenge
            Goldilocks3GPU::op_gpu(args_[i_args], &tmp3[args_[i_args + 1] * blockDim.x * FIELD_EXTENSION], &stepPointers_d->challenges_d[args_[i_args + 2] * FIELD_EXTENSION * blockDim.x], &stepPointers_d->challenges_d[args_[i_args + 3] * FIELD_EXTENSION * blockDim.x]);
            i_args += 4;
            break;
        }
        case 74:
        {
            // COPY eval to tmp3
            Goldilocks3GPU::copy_gpu(&tmp3[args_[i_args] * blockDim.x * FIELD_EXTENSION], &stepPointers_d->evals_d[args_[i_args + 1] * FIELD_EXTENSION * blockDim.x]);
            i_args += 2;
            break;
        }
        case 75:
        {
            // MULTIPLICATION WITH DEST: tmp3 - SRC0: eval - SRC1: challenge
            Goldilocks3GPU::mul_gpu(&tmp3[args_[i_args + 1] * blockDim.x * FIELD_EXTENSION], &stepPointers_d->evals_d[args_[i_args + 2] * FIELD_EXTENSION * blockDim.x], &stepPointers_d->challenges_d[args_[i_args + 3] * FIELD_EXTENSION * blockDim.x], &stepPointers_d->challenges_ops_d[args_[i_args + 3] * FIELD_EXTENSION * blockDim.x]);
            i_args += 4;
            break;
        }
        case 76:
        {
            // OPERATION WITH DEST: tmp3 - SRC0: challenge - SRC1: eval
            Goldilocks3GPU::op_gpu(args_[i_args], &tmp3[args_[i_args + 1] * blockDim.x * FIELD_EXTENSION], &stepPointers_d->challenges_d[args_[i_args + 2] * FIELD_EXTENSION * blockDim.x], &stepPointers_d->evals_d[args_[i_args + 3] * FIELD_EXTENSION * blockDim.x]);
            i_args += 4;
            break;
        }
        case 77:
        {
            // OPERATION WITH DEST: tmp3 - SRC0: tmp3 - SRC1: eval
            Goldilocks3GPU::op_gpu(args_[i_args], &tmp3[args_[i_args + 1] * blockDim.x * FIELD_EXTENSION], &tmp3[args_[i_args + 2] * blockDim.x * FIELD_EXTENSION], &stepPointers_d->evals_d[args_[i_args + 3] * FIELD_EXTENSION * blockDim.x]);
            i_args += 4;
            break;
        }
        case 78:
        {
            // OPERATION WITH DEST: tmp3 - SRC0: eval - SRC1: commit1
            Goldilocks3GPU::op_31_gpu(args_[i_args], &tmp3[args_[i_args + 1] * blockDim.x * FIELD_EXTENSION], &stepPointers_d->evals_d[args_[i_args + 2] * FIELD_EXTENSION * blockDim.x], &bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 3]] + args_[i_args + 4]) * blockDim.x]);
            i_args += 5;
            break;
        }
        case 79:
        {
            // OPERATION WITH DEST: tmp3 - SRC0: commit3 - SRC1: eval
            Goldilocks3GPU::op_gpu(args_[i_args], &tmp3[args_[i_args + 1] * blockDim.x * FIELD_EXTENSION], &bufferT_[(stepPointers_d->nColsStagesAcc_d[args_[i_args + 2]] + args_[i_args + 3]) * blockDim.x], &stepPointers_d->evals_d[args_[i_args + 4] * FIELD_EXTENSION * blockDim.x]);
            i_args += 5;
            break;
        }
        default:
        {
            return;
        }
        }
    }
}

__global__ void _transposeToBuffer(StepsPointers *stepPointers_d, uint64_t row, uint32_t stage, bool domainExtended, uint32_t istream){

    gl64_t *bufferT_ = &(stepPointers_d->bufferT_d[istream * stepPointers_d->dimBufferT]);
    gl64_t *bufferConsts_ = &(stepPointers_d->bufferConsts_d[istream * stepPointers_d->dimBufferConsts]);
    gl64_t *bufferPols_ = &(stepPointers_d->bufferPols_d[istream * stepPointers_d->dimBufferPols]);
    gl64_t * x = domainExtended ? stepPointers_d->x_2ns_d : stepPointers_d->x_n_d;

    uint64_t nextStrides[2] = {0, stepPointers_d->nextStride};
    for (uint64_t o = 0; o < 2; ++o)
    {
        for (uint64_t k = 0; k < stepPointers_d->nConstants; ++k)
        {
        
            bufferT_[(stepPointers_d->nColsStagesAcc_d[5 * o] + k) * blockDim.x + threadIdx.x] = bufferConsts_[(threadIdx.x+nextStrides[o])* stepPointers_d->nConstants + k]; 
        }
    }

    bufferT_[stepPointers_d->nConstants * blockDim.x + threadIdx.x] = x[row + threadIdx.x];
    bufferT_[(stepPointers_d->nConstants + 1) * blockDim.x + threadIdx.x] = stepPointers_d->zi_d[row + threadIdx.x];

    uint32_t offset_pols = 0;
    uint64_t nStages = 3;
    for (uint64_t s = 1; s <= nStages; ++s) 
    {
        for (uint64_t o = 0; o < 2; ++o) 
        {
            for (uint64_t k = 0; k < stepPointers_d->nColsStages_d[s]; ++k)
            {    
                uint64_t l = threadIdx.x + nextStrides[o];
                bufferT_[(stepPointers_d->nColsStagesAcc_d[5 * o + s] + k) * blockDim.x + threadIdx.x] = bufferPols_[offset_pols + l * stepPointers_d->nColsStages_d[s] + k];
            }
        }
        offset_pols += stepPointers_d->nColsStages_d[s] * (blockDim.x+stepPointers_d->nextStride);
    }
    if (stage == 5)
    {
        
        for (uint64_t o = 0; o < 2; ++o)
        {
            for (uint64_t k = 0; k < stepPointers_d->nColsStages_d[nStages + 1]; ++k)
            {
                uint64_t l = threadIdx.x + nextStrides[o];
                bufferT_[(stepPointers_d->nColsStagesAcc_d[5 * o + nStages + 1] + k) * blockDim.x + threadIdx.x] = bufferPols_[offset_pols + l * stepPointers_d->nColsStages_d[nStages + 1] + k];
            }
        }

        for (uint64_t d = 0; d < 2; ++d)
        {
            for (uint64_t i = 0; i < FIELD_EXTENSION; ++i)
            {
                bufferT_[(stepPointers_d->nColsStagesAcc_d[11] + FIELD_EXTENSION * d + i) * blockDim.x + threadIdx.x] = stepPointers_d->xDivXSubXi_d[(d * stepPointers_d->domainSize + row + threadIdx.x)*FIELD_EXTENSION+i];
            }
        }
    }
     
}

__global__ void _transposeFromBuffer(StepsPointers *stepPointers_d, uint64_t row, uint32_t stage, bool domainExtended, uint32_t istream){

    gl64_t *bufferT_ = &(stepPointers_d->bufferT_d[istream * stepPointers_d->dimBufferT]);

    if (domainExtended)
    {
        gl64_t *bufferPols_ = &(stepPointers_d->bufferPols_d[(istream+1) * stepPointers_d->dimBufferPols-stepPointers_d->nColsStages_d[10]*blockDim.x]);
        // Store either polinomial f or polinomial q
        for (uint64_t k = 0; k < stepPointers_d->nColsStages_d[10]; ++k)
        {
            bufferPols_[threadIdx.x*stepPointers_d->nColsStages_d[10]+k] = bufferT_[(stepPointers_d->nColsStagesAcc_d[10] + k) * blockDim.x + threadIdx.x];
        }
    }else{
        gl64_t *bufferPols_ = &(stepPointers_d->bufferPols_d[(istream) * stepPointers_d->dimBufferPols]);
        uint64_t nStages = 3;
        uint64_t offset_pols_d=0;
        for (uint64_t s = 2; s <= nStages + 1; ++s)
        {
            gl64_t *buffT = &bufferT_[stepPointers_d->nColsStagesAcc_d[s]*blockDim.x];
            for (uint64_t k = 0; k < stepPointers_d->nColsStages_d[s]; ++k)
            {
                bufferPols_[offset_pols_d +  threadIdx.x *stepPointers_d->nColsStages_d[s]+k] = buffT[k*blockDim.x + threadIdx.x];
            }
            offset_pols_d += stepPointers_d->nColsStages_d[s]*blockDim.x;
        }
    }
}
#endif
